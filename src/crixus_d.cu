#include "hip/hip_runtime.h"
#ifndef CRIXUS_D_CU
#define CRIXUS_D_CU

#include <hip/hip_runtime.h>
#include "lock.cuh"
#include "crixus_d.cuh"

__global__ void set_bound_elem (uf4 *pos, uf4 *norm, float *surf, ui4 *ep, unsigned int nbe, float *xminp, float *xminn, float *nminp, float*nminn, Lock lock, int nvert)
{
	float ddum[3];
	unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
	__shared__ float xminp_c[threadsPerBlock];
	__shared__ float xminn_c[threadsPerBlock];
	__shared__ float nminp_c[threadsPerBlock];
	__shared__ float nminn_c[threadsPerBlock];
	float xminp_t;
	float xminn_t;
	float nminp_t;
	float nminn_t;
	int i_c = threadIdx.x;
	xminp_t = *xminp;
	xminn_t = *xminn;
	nminp_t = *nminp;
	nminn_t = *nminn;
	while(i<nbe){
		//formula: a = 1/4 sqrt(4*a^2*b^2-(a^2+b^2-c^2)^2)
		float a2 = 0.;
		float b2 = 0.;
		float c2 = 0.;
		ddum[0] = 0.;
		ddum[1] = 0.;
		ddum[2] = 0.;
		for(unsigned int j=0; j<3; j++){
			ddum[j] += pos[ep[i].a[0]].a[j]/3.;
			ddum[j] += pos[ep[i].a[1]].a[j]/3.;
			ddum[j] += pos[ep[i].a[2]].a[j]/3.;
			a2 += pow(pos[ep[i].a[0]].a[j]-pos[ep[i].a[1]].a[j],2);
			b2 += pow(pos[ep[i].a[1]].a[j]-pos[ep[i].a[2]].a[j],2);
			c2 += pow(pos[ep[i].a[2]].a[j]-pos[ep[i].a[0]].a[j],2);
		}
		if(norm[i].a[2] > 1e-5 && xminp_t > ddum[2]){
			xminp_t = ddum[2];
			nminp_t = norm[i].a[2];
		}
		if(norm[i].a[2] < -1e-5 && xminn_t > ddum[2]){
			xminn_t = ddum[2];
			nminn_t = norm[i].a[2];
		}
		surf[i] = 0.25*sqrt(4.*a2*b2-pow(a2+b2-c2,2));
    for(int j=0; j<3; j++)
		  pos[i+nvert].a[j] = ddum[j];
		i += blockDim.x*gridDim.x;
	}

	xminp_c[i_c] = xminp_t;
	xminn_c[i_c] = xminn_t;
	nminp_c[i_c] = nminp_t;
	nminn_c[i_c] = nminn_t;
	__syncthreads();

	int j = blockDim.x/2;
	while (j!=0){
		if(i_c < j){
			if(xminp_c[i_c+j] < xminp_c[i_c]){
				xminp_c[i_c] = xminp_c[i_c+j];
				nminp_c[i_c] = nminp_c[i_c+j];
			}
			if(xminn_c[i_c+j] < xminn_c[i_c]){
				xminn_c[i_c] = xminn_c[i_c+j];
				nminn_c[i_c] = nminn_c[i_c+j];
			}
		}
		__syncthreads();
		j /= 2;
	}

	if(i_c == 0){
		lock.lock();
		if(xminp_c[0] < *xminp){
			*xminp = xminp_c[0];
			*nminp = nminp_c[0];
		}
		if(xminn_c[0] < *xminn){
			*xminn = xminn_c[0];
			*nminn = nminn_c[0];
		}
		lock.unlock();
	}
}


#endif
