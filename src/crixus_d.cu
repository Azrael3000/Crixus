#include "hip/hip_runtime.h"
#ifndef CRIXUS_D_CU
#define CRIXUS_D_CU

#include <hip/hip_runtime.h>
#include "lock.cuh"
#include "crixus_d.cuh"
#include "return.h"
#include "crixus.h"

__global__ void set_bound_elem (uf4 *pos, uf4 *norm, float *surf, ui4 *ep, unsigned int nbe, float *xminp, float *xminn, float *nminp, float*nminn, Lock lock, int nvert)
{
	float ddum[3];
	unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
	__shared__ float xminp_c[threadsPerBlock];
	__shared__ float xminn_c[threadsPerBlock];
	__shared__ float nminp_c[threadsPerBlock];
	__shared__ float nminn_c[threadsPerBlock];
	float xminp_t;
	float xminn_t;
	float nminp_t;
	float nminn_t;
	int i_c = threadIdx.x;
	xminp_t = *xminp;
	xminn_t = *xminn;
	nminp_t = *nminp;
	nminn_t = *nminn;
	while(i<nbe){
		//formula: a = 1/4 sqrt(4*a^2*b^2-(a^2+b^2-c^2)^2)
		float a2 = 0.;
		float b2 = 0.;
		float c2 = 0.;
		ddum[0] = 0.;
		ddum[1] = 0.;
		ddum[2] = 0.;
		for(unsigned int j=0; j<3; j++){
			ddum[j] += pos[ep[i].a[0]].a[j]/3.;
			ddum[j] += pos[ep[i].a[1]].a[j]/3.;
			ddum[j] += pos[ep[i].a[2]].a[j]/3.;
			a2 += pow(pos[ep[i].a[0]].a[j]-pos[ep[i].a[1]].a[j],2);
			b2 += pow(pos[ep[i].a[1]].a[j]-pos[ep[i].a[2]].a[j],2);
			c2 += pow(pos[ep[i].a[2]].a[j]-pos[ep[i].a[0]].a[j],2);
		}
		if(norm[i].a[2] > 1e-5 && xminp_t > ddum[2]){
			xminp_t = ddum[2];
			nminp_t = norm[i].a[2];
		}
		if(norm[i].a[2] < -1e-5 && xminn_t > ddum[2]){
			xminn_t = ddum[2];
			nminn_t = norm[i].a[2];
		}
		surf[i] = 0.25*sqrt(4.*a2*b2-pow(a2+b2-c2,2));
    for(int j=0; j<3; j++)
		  pos[i+nvert].a[j] = ddum[j];
		i += blockDim.x*gridDim.x;
	}

	xminp_c[i_c] = xminp_t;
	xminn_c[i_c] = xminn_t;
	nminp_c[i_c] = nminp_t;
	nminn_c[i_c] = nminn_t;
	__syncthreads();

	int j = blockDim.x/2;
	while (j!=0){
		if(i_c < j){
			if(xminp_c[i_c+j] < xminp_c[i_c]){
				xminp_c[i_c] = xminp_c[i_c+j];
				nminp_c[i_c] = nminp_c[i_c+j];
			}
			if(xminn_c[i_c+j] < xminn_c[i_c]){
				xminn_c[i_c] = xminn_c[i_c+j];
				nminn_c[i_c] = nminn_c[i_c+j];
			}
		}
		__syncthreads();
		j /= 2;
	}

	if(i_c == 0){
		lock.lock();
		if(xminp_c[0] < *xminp){
			*xminp = xminp_c[0];
			*nminp = nminp_c[0];
		}
		if(xminn_c[0] < *xminn){
			*xminn = xminn_c[0];
			*nminn = nminn_c[0];
		}
		lock.unlock();
	}
}

__global__ void swap_normals (uf4 *norm, int nbe)
{
	unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
	while(i<nbe){
    for(int j=0; j<3; j++)
		  norm[i].a[j] *= -1.;
		i += blockDim.x*gridDim.x;
	}
}

__global__ void find_links(uf4 *pos, int nvert, uf4 *dmax, uf4 *dmin, float dr, int *newlink, int idim)
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	while(i<nvert){
		if(fabs(pos[i].a[idim]-(*dmax).a[idim])<1e-5*dr){
			for(unsigned int j=0; j<nvert; j++){
				if(j==i) continue;
				if(sqrt(pow(pos[i].a[(idim+1)%3]-pos[j].a[(idim+1)%3],(float)2.)+ \
				        pow(pos[i].a[(idim+2)%3]-pos[j].a[(idim+2)%3],(float)2.)+ \
								pow(pos[j].a[idim      ]- (*dmin).a[idim]      ,(float)2.) ) < 1e-4*dr){
					newlink[i] = j;
					//"delete" vertex
					for(int k=0; k<3; k++)
						pos[i].a[k] = -1e10;
					break;
				}
				if(j==nvert-1){
					// cout << " [FAILED]" << endl;
					return; //NO_PER_VERT;
				}
			}
		}
		i += blockDim.x*gridDim.x;
	}
}

//__device__ volatile int lock_per_mutex[2]={0,0};
__global__ void periodicity_links (uf4 *pos, ui4 *ep, int nvert, int nbe, uf4 *dmax, uf4 *dmin, float dr, int *newlink, int idim)
{
	//find corresponding vertices
	unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;

	//relink
	i = blockIdx.x*blockDim.x+threadIdx.x;
	while(i<nbe){
    for(int j=0; j<3; j++){
		  if(newlink[ep[i].a[j]] != -1)
        ep[i].a[j] = newlink[ep[i].a[j]];
    }
		i += blockDim.x*gridDim.x;
	}

	return;
}

__global__ void calc_trisize(ui4 *ep, int *trisize, int nbe)
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	while(i<nbe){
		for(unsigned int j=0; j<3; j++){
			atomicAdd(&trisize[ep[i].a[j]],1);
		}
		i += blockDim.x*gridDim.x;
	}
}

//__device__ volatile int lock_mutex[2];
#ifndef bdebug
__global__ void calc_vert_volume (uf4 *pos, uf4 *norm, ui4 *ep, float *vol, int *trisize, uf4 *dmin, uf4 *dmax, int nvert, int nbe, float dr, float eps, bool *per)
#else
__global__ void calc_vert_volume (uf4 *pos, uf4 *norm, ui4 *ep, float *vol, int *trisize, uf4 *dmin, uf4 *dmax, int nvert, int nbe, float dr, float eps, bool *per, uf4 *debug, float* debugp)
#endif
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;

	//sort neighbouring vertices
	//calculate volume (geometry factor)
	unsigned int gsize = gres*2+1; //makes sure that grid is large enough
	float gdr = dr/(float)gres;
	float vgrid;
	float cvec[trimax][12][3];
	int tri[trimax][3];
	float avnorm[3];
	bool first[trimax];
	uf4 edgen[trimax];
	float vnorm;
	bool closed;
	int iduma[3];
	float sp;

	i = blockIdx.x*blockDim.x+threadIdx.x;
	while(i<nvert){

		//vertex has been deleted
		if(pos[i].a[0] < -1e9){
			i += blockDim.x*gridDim.x;
			continue;
		}

		//initialize variables
		closed = true;
		vol[i] = 0.;
		unsigned int tris = trisize[i];
    if(tris > trimax)
      return; //exception needs to be thrown
		for(unsigned int j=0; j<tris; j++){
			first[j] = true;
			for(unsigned int k=0; k<4; k++)
				edgen[j].a[k] = 0.;
		}
		for(unsigned int j=0; j<3; j++)
      avnorm[j] = 0.;

		//find connected faces
		unsigned int itris = 0;
		for(unsigned int j=0; j<nbe; j++){
			for(unsigned int k=0; k<3; k++){
				if(ep[j].a[k] == i){
					tri[itris][0] = ep[j].a[(k+1)%3];
					tri[itris][1] = ep[j].a[(k+2)%3];
					tri[itris][2] = j;
#ifdef bdebug
//				for(int j=0; j<tris; j++){
					if(i==bdebug){
					debugp[4+itris*4+0] = ep[j].a[0]+960;
					debugp[4+itris*4+1] = ep[j].a[1]+960;
					debugp[4+itris*4+2] = ep[j].a[2]+960;
					debugp[4+itris*4+3] = tri[itris][2]+2498;
				}
//				}
#endif
					itris++;
				}
			}
		}

		//try to put neighbouring faces next to each other
		for(unsigned int j=0; j<tris; j++){
			for(unsigned int k=j+1; k<tris; k++){
				if(tri[j][1] == tri[k][0]){
					if(k!=j+1){
						for(int l=0; l<3; l++){
							iduma[l] = tri[j+1][l];
							tri[j+1][l] = tri[k][l];
							tri[k][l] = iduma[l];
						}
					}
					break;
				}
				if(tri[j][1] == tri[k][1]){
					iduma[0] = tri[k][1];
					iduma[1] = tri[k][0];
					iduma[2] = tri[k][2];
					for(int l=0; l<3; l++){
						tri[k][l] = tri[j+1][l];
						tri[j+1][l] = iduma[l];
					}
					break;
				}
				if(k==tris-1) closed = false;
			}
		}
		if(tri[0][0] != tri[tris-1][1]){
			closed = false;
		}
		
		// calculate average normal at edge
		itris = 0;
		for(unsigned int j=0; j<nbe; j++){
			for(unsigned int k=0; k<tris; k++){
				if((int)(edgen[k].a[3]+eps)==2)
					continue;
				int vfound = 0;
				for(unsigned int l=0; l<3; l++){
					if(ep[j].a[l] == tri[k][0] || ep[j].a[l] == tri[k][1])
						vfound++;
				}
				if(vfound==2){
					for(unsigned int l=0; l<3; l++)
						edgen[k].a[l] += norm[j].a[l];
					edgen[k].a[3]+=1.;
				}
				if((int)(edgen[k].a[3]+eps)==2){ //cross product to determine normal of wall
					float tmpvec[3], edge[3];
					for(unsigned int n=0; n<3; n++) edge[n] = pos[tri[k][0]].a[n] - pos[tri[k][1]].a[n];
					for(unsigned int n=0; n<3; n++)	tmpvec[n] = edgen[k].a[(n+1)%3]*edge[(n+2)%3]-edgen[k].a[(n+2)%3]*edge[(n+1)%3];
					for(unsigned int n=0; n<3; n++) edgen[k].a[n] = tmpvec[n];
				}
			}
		}

#ifdef bdebug
			if(i==bdebug){
//				for(int j=0; j<100; j++) debugp[j] = 0.;
				debugp[0] = tris;
				debugp[1] = pos[i].a[0];
				debugp[2] = pos[i].a[1];
				debugp[3] = pos[i].a[2];
			}
#endif

		//start big loop over all numerical integration points
		for(unsigned int k=0; k<gsize; k++){
		for(unsigned int l=0; l<gsize; l++){
		for(unsigned int m=0; m<gsize; m++){

			float gp[3]; //gridpoint in coordinates relative to vertex
			gp[0] = (((float)k-(float)(gsize-1)/2))*gdr;
			gp[1] = (((float)l-(float)(gsize-1)/2))*gdr;
			gp[2] = (((float)m-(float)(gsize-1)/2))*gdr;
			vgrid = 0.;

#ifdef bdebug
			if(i==bdebug){
			for(int j=0; j<3; j++) debug[k+l*gsize+m*gsize*gsize].a[j] = gp[j] + pos[i].a[j];
			debug[k+l*gsize+m*gsize*gsize].a[3] = -1.;
			}
#endif

			//create cubes
			for(unsigned int j=0; j<tris; j++){
				if(k+l+m==0){
					//setting up cube directions
					for(unsigned int n=0; n<3; n++) cvec[j][2][n] = norm[tri[j][2]].a[n]; //normal of boundary element
					vnorm = 0.;
					for(unsigned int n=0; n<3; n++){
						cvec[j][0][n] = pos[tri[j][0]].a[n]-pos[i].a[n]; //edge 1
						if(per[n]&&fabs(cvec[j][0][n])>2*dr)	cvec[j][0][n] += sgn(cvec[j][0][n])*(-(*dmax).a[n]+(*dmin).a[n]); //periodicity
						vnorm += pow(cvec[j][0][n],2);
					}
					vnorm = sqrt(vnorm);
					for(unsigned int n=0; n<3; n++) cvec[j][0][n] /= vnorm;
					for(unsigned int n=0; n<3; n++)	cvec[j][1][n] = cvec[j][0][(n+1)%3]*cvec[j][2][(n+2)%3]-cvec[j][0][(n+2)%3]*cvec[j][2][(n+1)%3]; //cross product of normal and edge1
					vnorm = 0.;
					for(unsigned int n=0; n<3; n++){
						cvec[j][3][n] = pos[tri[j][1]].a[n]-pos[i].a[n]; //edge 2
						if(per[n]&&fabs(cvec[j][3][n])>2*dr)	cvec[j][3][n] += sgn(cvec[j][3][n])*(-(*dmax).a[n]+(*dmin).a[n]); //periodicity
						vnorm += pow(cvec[j][3][n],2);
						avnorm[n] -= norm[tri[j][2]].a[n];
					}
					vnorm = sqrt(vnorm);
					for(unsigned int n=0; n<3; n++) cvec[j][3][n] /= vnorm;
					for(unsigned int n=0; n<3; n++)	cvec[j][4][n] = cvec[j][3][(n+1)%3]*cvec[j][2][(n+2)%3]-cvec[j][3][(n+2)%3]*cvec[j][2][(n+1)%3]; //cross product of normal and edge2
				}
				//filling vgrid
				bool incube[5] = {false, false, false, false, false};
				for(unsigned int n=0; n<5; n++){
					sp = 0.;
					for(unsigned int o=0; o<3; o++) sp += gp[o]*cvec[j][n][o];
					if(fabs(sp)<=dr/2.+eps) incube[n] = true;
				}
				if((incube[0] && incube[1] && incube[2]) || (incube[2] && incube[3] && incube[4])){
					vgrid = 1.;
#ifdef bdebug
			if(i==bdebug) debug[k+l*gsize+m*gsize*gsize].a[3] = 1.;
#endif
					if(k+l+m!=0) break; //makes sure that in the first grid point we loop over all triangles j s.t. values are initialized correctly.
				}
			}
			//end create cubes

			//remove points based on planes (voronoi diagram & walls)
			float tvec[3][3];
			for(unsigned int j=0; j<tris; j++){
				if(vgrid<eps) break; //gridpoint already empty
				if(first[j]){
					first[j] = false;
					//set up plane normals and points
					for(unsigned int n=0; n<3; n++){
						cvec[j][5][n] = pos[tri[j][0]].a[n]-pos[i].a[n]; //normal of plane voronoi
						if(per[n]&&fabs(cvec[j][5][n])>2*dr)	cvec[j][5][n] += sgn(cvec[j][5][n])*(-(*dmax).a[n]+(*dmin).a[n]); //periodicity
						cvec[j][6][n] = pos[i].a[n]+cvec[j][5][n]/2.; //position of plane voronoi
						tvec[0][n] = cvec[j][5][n]; // edge 1
						tvec[1][n] = pos[tri[j][1]].a[n]-pos[i].a[n]; // edge 2
						if(per[n]&&fabs(tvec[1][n])>2*dr)	tvec[1][n] += sgn(tvec[1][n])*(-(*dmax).a[n]+(*dmin).a[n]); //periodicity
						if(!closed){
							cvec[j][7][n] = tvec[1][n]; //normal of plane voronoi 2
							cvec[j][8][n] = pos[i].a[n]+cvec[j][7][n]/2.; //position of plane voronoi 2
						}
						tvec[2][n] = avnorm[n]; // negative average normal
					}
					for(unsigned int n=0; n<3; n++){
						for(unsigned int k=0; k<3; k++){
							cvec[j][k+9][n] = tvec[k][(n+1)%3]*tvec[(k+1)%3][(n+2)%3]-tvec[k][(n+2)%3]*tvec[(k+1)%3][(n+1)%3]; //normals of tetrahedron planes
						}
					}
					sp = 0.;
					for(unsigned int n=0; n<3; n++) sp += norm[tri[j][2]].a[n]*cvec[j][9][n]; //test whether normals point inward tetrahedron, if no flip normals
					if(sp > 0.){
						for(unsigned int k=0; k<3; k++){
							for(unsigned int n=0; n<3; n++)	cvec[j][k+9][n] *= -1.;
						}
					}
					//edge normal to point in right direction
					sp = 0.;
					for(unsigned int n=0; n<3; n++) sp += edgen[j].a[n]*cvec[j][5][n]; //sp of edge plane normal and vector pointing from vertex to plane point
					if(sp < 0.){
						for(unsigned int n=0; n<3; n++) edgen[j].a[n] *= -1.; //flip
					}
#ifdef bdebug
//				for(int j=0; j<tris; j++){
					if(i==bdebug){
					debugp[4+j*4+0] = edgen[j].a[0];
					debugp[4+j*4+1] = edgen[j].a[1];
					debugp[4+j*4+2] = edgen[j].a[2];
					debugp[4+j*4+3] = tri[j][2]+2498;
					}
//				}
#endif
				}

			  //remove unwanted points and sum up for volume
				//voronoi plane
				for(unsigned int n=0; n<3; n++) tvec[0][n] = gp[n] + pos[i].a[n] - cvec[j][6][n];
				sp = 0.;
				for(unsigned int n=0; n<3; n++) sp += tvec[0][n]*cvec[j][5][n];
				if(sp>0.+eps){
					vgrid = 0.;
#ifdef bdebug
			if(i==bdebug) debug[k+l*gsize+m*gsize*gsize].a[3] = 0.;
#endif
					break;
				}
				else if(fabs(sp) < eps){
					vgrid /= 2.;
				}
				//voronoi plane 2
				if(!closed){
					for(unsigned int n=0; n<3; n++) tvec[0][n] = gp[n] + pos[i].a[n] - cvec[j][8][n];
					sp = 0.;
					for(unsigned int n=0; n<3; n++) sp += tvec[0][n]*cvec[j][7][n];
					if(sp>0.+eps){
						vgrid = 0.;
						break;
					}
					else if(fabs(sp) < eps){
						vgrid /= 2.;
					}
				}
				//walls
				bool half = false;
				for(unsigned int o=0; o<3; o++){
					sp = 0.;
					for(unsigned int n=0; n<3; n++) sp += gp[n]*cvec[j][9+o][n];
					if(sp<0.-eps) break;
					if(fabs(sp)<eps && o==0) half=true;
					if(o==2 && !half){
						vgrid = 0.;
#ifdef bdebug
			if(i==bdebug) debug[k+l*gsize+m*gsize*gsize].a[3] = 0.;
#endif
						break;
					}
					else if(o==2 && half){
						vgrid /= 2.;
					}
				}
				//edges
				sp = 0.;
				for(unsigned int n=0; n<3; n++){
					tvec[0][n] = gp[n] - cvec[j][5][n];
					sp += tvec[0][n]*edgen[j].a[n];
				}
				if(sp>0.+eps){
					vgrid = 0.;
#ifdef bdebug
			if(i==bdebug) debug[k+l*gsize+m*gsize*gsize].a[3] = -0.5;
#endif
					break;
				}
				else if(fabs(sp) < eps){
					vgrid /= 2.;
				}
				if(vgrid < eps) break;

				//volume sum
				if(j==tris-1)	vol[i] += vgrid;
			}

		}
		}
		}
		//end looping through all gridpoints

		//calculate volume
		vol[i] *= pow(dr/(float)gres,3);

		i += blockDim.x*gridDim.x;
	}
}

__global__ void fill_fluid (uf4 *fpos, float xmin, float xmax, float ymin, float ymax, float zmin, float zmax, float eps, float dr, int *nfib, int fmax, Lock lock)
{
	//this can be a bit more complex in order to fill complex geometries
	__shared__ int nfib_cache[threadsPerBlock];
	int idim = (floor((ymax+eps-ymin)/dr)+1)*(floor((xmax+eps-xmin)/dr)+1);
	int jdim =  floor((xmax+eps-xmin)/dr)+1;
	int i, j, k, tmp, nfib_tmp;
	int tid = threadIdx.x;

	nfib_tmp = 0;
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	while(id<fmax){
		k = id/idim;
		tmp = id%idim;
		j = tmp/jdim;
		i = tmp%jdim;
		fpos[id].a[0] = xmin + (float)i*dr;
		fpos[id].a[1] = ymin + (float)j*dr;
		fpos[id].a[2] = zmin + (float)k*dr;
		nfib_tmp++;
		//if position should not be filled use a[0] = -1e10 and do not increment nfib_tmp
		id += blockDim.x*gridDim.x;
	}
	nfib_cache[tid] = nfib_tmp;

	__syncthreads();

	j = blockDim.x/2;
	while (j!=0){
		if(tid < j)
			nfib_cache[tid] += nfib_cache[tid+j];
		__syncthreads();
		j /= 2;
	}

	if(tid == 0){
		lock.lock();
		*nfib += nfib_cache[0];
		lock.unlock();
	}
}

#endif
