#include "hip/hip_runtime.h"
/***********************************\
 *
 * TODO LIST:
 * - Version 0.6:
 *   - in/outflow/moving boundary option
 *   - read/write config file
 *   - replace uf4 by uf3 (class with float[3])
 *   - do more calculations using the vector_math.h library
 *   - while doing calculations on kernel check
 *     files on host and maybe read them already
 *   - clean up code (free norm, ep when they
 *     are no longer needed and not at the end)
 *   - refactor the codes so that crixus.cu is split into functions
 * - Version 0.7:
 *   - CSG for volume computation
 *
\***********************************/

#ifndef CRIXUS_CU
#define CRIXUS_CU

#include "crixus.h"
#include "return.h"
#include "crixus_d.cuh"
#include "lock.cuh"
#include "vector_math.h"

using namespace std;

int crixus_main(int argc, char** argv){
  //host
  cout << endl;
  cout << "\t**************************************" << endl;
  cout << "\t*                                    *" << endl;
  cout << "\t*             C R I X U S            *" << endl;
  cout << "\t*                                    *" << endl;
  cout << "\t**************************************" << endl;
  cout << "\t* Version     : 0.5                  *" << endl;
  cout << "\t* Date        : 27.03.2014           *" << endl;
  cout << "\t* Author      : Arno Mayrhofer       *" << endl;
  cout << "\t* Contributors: Christophe Kassiotis *" << endl;
  cout << "\t*               F-X Morel            *" << endl;
  cout << "\t*               Martin Ferrand       *" << endl;
  cout << "\t*               Agnes Leroy          *" << endl;
  cout << "\t*               Antoine Joly         *" << endl;
  cout << "\t*               Giuseppe Bilotta     *" << endl;
  cout << "\t**************************************" << endl;
  cout << endl;
  float m_v_floats[12];
  unsigned int through;
  short attribute;
  unsigned int num_of_facets;
  const unsigned int bitPerUint = 8*sizeof(unsigned int);

  if(argc==1){
    cout << "No configuration file specified." << endl;
    cout << "Correct use: crixus filename" << endl;
    cout << "Example use: crixus box.ini" << endl;
    return NO_FILE;
  }
  else if(argc>2){
    cout << "Ignoring additional arguments after configuration file." << endl;
  }
  string configfname = argv[1];
  INIReader config(configfname);

  if (config.ParseError() < 0) {
    std::cout << "Can't load configuration file " << configfname << endl;;
    return CANT_READ_CONFIG;
  }
  string fname = config.Get("mesh", "stlfile", "UNKNOWN");

  //looking for cuda devices without timeout
  cout << "Selecting GPU ...";
  int dcount, maxblock=0, maxthread;
  maxthread = threadsPerBlock;
  bool found = false;
  CUDA_SAFE_CALL( hipGetDeviceCount(&dcount) );
  for (int i=0; i<dcount; i++){
    hipDeviceProp_t prop;
    CUDA_SAFE_CALL( hipGetDeviceProperties(&prop,i) );
    if(!prop.kernelExecTimeoutEnabled){
      found = true;
      CUDA_SAFE_CALL( hipSetDevice(i) );
      maxthread = prop.maxThreadsPerBlock;
      maxblock  = prop.maxGridSize[0];
      cout << " Id: " << i << " (" << maxthread << ", " << maxblock << ") ...";
      if(maxthread < threadsPerBlock){
        cout << " [FAILED]" << endl;
        return MAXTHREAD_TOO_BIG;
      }
      cout << " [OK]" << endl;
      break;
    }
  }
  if(!found){
    hipDeviceProp_t prop;
    CUDA_SAFE_CALL( hipGetDeviceProperties(&prop,0) );
    CUDA_SAFE_CALL( hipSetDevice(0) );
    maxthread = prop.maxThreadsPerBlock;
    maxblock  = prop.maxGridSize[0];
    cout << " Id: " << 0 << " (" << maxthread << ", " << maxblock << ") ...";
    if(maxthread < threadsPerBlock){
      cout << " [FAILED]" << endl;
      return MAXTHREAD_TOO_BIG;
    }
    cout << " [OK]" << endl;
    cout << "\n\tWARNING:" << endl;
    cout << "\tCould not find GPU without timeout." << endl;
    cout << "\tIf execution terminates with timeout reduce gres.\n" << endl;
  }
  if(maxthread != threadsPerBlock){
    cout << "\n\tINFORMATION:" << endl;
    cout << "\tthreadsPerBlock is not equal to maximum number of available threads.\n" << endl;
  }

  //Reading file
  cout << "Opening file " << fname << " ...";
  ifstream stl_file (fname.c_str(), ios::in);
  if(!stl_file.is_open()){
    cout << " [FAILED]" << endl;
    return FILE_NOT_OPEN;
  }
  cout << " [OK]" << endl;

  float dr = config.GetReal("mesh", "dr", -1);
  cout << "Mesh size: " << dr << endl;

  cout << "Checking whether stl file is not ASCII ...";
  fflush(stdout);
  bool issolid = true;
  char header[6] = "solid";
  for (int i=0; i<5; i++){
    char dum;
    stl_file.read((char *)&dum, sizeof(char));
    if(dum!=header[i]){
      issolid = false;
      break;
    }
  }
  if(issolid){
    cout << " [FAILED]" << endl;
    stl_file.close();
    return STL_NOT_BINARY;
  }
  stl_file.close();
  cout << " [OK]" << endl;

  // reopen file in binary mode
  stl_file.open(fname.c_str(), ios::in | ios::binary);

  // read header
  for (int i=0; i<20; i++){
    float dum;
    stl_file.read((char *)&dum, sizeof(float));
  }
  // get number of facets
  stl_file.read((char *)&num_of_facets, sizeof(int));
  cout << "Reading " << num_of_facets << " facets ...";
  fflush(stdout);

  // define variables
  vector< vector<float> > pos;
  vector< vector<float> > norm;
  vector< vector<float> >::iterator it, jt;
  vector< vector<unsigned int> > epv;
  unsigned int nvert, nbe;
  vector<unsigned int> idum;
  vector<float> ddum;
  ddum.resize(3, 0.0);
  idum.resize(3, 0);

  // read data
  through = 0;
  float xmin = 1e10, xmax = -1e10;
  float ymin = 1e10, ymax = -1e10;
  float zmin = 1e10, zmax = -1e10;
  while ((through < num_of_facets) & (!stl_file.eof()))
  {
    for (int i=0; i<12; i++)
    {
      stl_file.read((char *)&m_v_floats[i], sizeof(float));
    }
    for(int i=0;i<3;i++) ddum[i] = (float)m_v_floats[i];
    norm.push_back(ddum);
    // save the three vertices in an array
    vector<float> tmp;
    tmp.resize(4, 0.0);
    vector< vector<float> > vdum;
    for(int j=0;j<3;j++){
      for(int i=0;i<3;i++){
        tmp[i] = (float)m_v_floats[i+3*(j+1)];
      }
      tmp[3] = (float)j + 0.5; // add 0.5 so that when we (int) cast we get the proper number
      vdum.push_back(tmp);
    }
    // loop over all existing vertices to see whether it already exists.
    int k = 0;
    for(it = pos.begin(); it < pos.end() && !vdum.empty(); it++){
      for(jt = vdum.begin(); jt < vdum.end(); ){
        //compute square distance between two particles
        float diff = 0.0;
        for(int i=0;i<3;i++) diff += ((*it)[i] - (*jt)[i])*((*it)[i] - (*jt)[i]);
        // if we are very far away we can see that after the first distance calculation
        // none will ever match
        if(diff > 5.0*dr*dr)
          break;
        else if(diff < 1e-5*dr*dr){
          int localVertIndex = (int)(*jt)[3];
          idum[localVertIndex] = k;
          vdum.erase(jt);
          break; // if we found one match, the others wont match (hopefully)
        }
        else
          ++jt;
      }
      k++;
    }
    // loop only over the remaining vertices that have not been found
    for(jt = vdum.begin(); jt < vdum.end(); jt++){
      for(int j=0; j<3; j++)
        ddum[j] = (*jt)[j];
      pos.push_back(ddum);
      xmin = (xmin > ddum[0]) ? ddum[0] : xmin;
      xmax = (xmax < ddum[0]) ? ddum[0] : xmax;
      ymin = (ymin > ddum[1]) ? ddum[1] : ymin;
      ymax = (ymax < ddum[1]) ? ddum[1] : ymax;
      zmin = (zmin > ddum[2]) ? ddum[2] : zmin;
      zmax = (zmax < ddum[2]) ? ddum[2] : zmax;
      int localVertIndex = (int)(*jt)[3];
      idum[localVertIndex] = pos.size() - 1;
    }
    vdum.clear();
    epv.push_back(idum);
    stl_file.read((char *)&attribute, sizeof(short));
    through++;
  }
  stl_file.close();
  if(num_of_facets != norm.size()){
    cout << " [FAILED]" << endl;
    return READ_ERROR;
  }
  nvert = pos.size();
  nbe   = norm.size();
  //create and copy vectors to arrays
  uf4 *norma, *posa;
  float *vola, *surf;
  ui4 *ep;
  norma = new uf4   [nbe];
  posa  = new uf4   [nvert+nbe];
  vola  = new float [nvert];
  surf  = new float [nbe]; //AM-TODO: could go to norma[3]
  ep    = new ui4   [nbe];
  for(unsigned int i=0; i<max(nvert,nbe); i++){
    if(i<nbe){
      for(int j=0; j<3; j++){
        norma[i].a[j] = norm[i][j];
        ep[i].a[j] = epv[i][j];
      }
    }
    if(i<nvert){
      for(int j=0; j<3; j++)
        posa[i].a[j] = pos[i][j];
      vola[i] = 0.;
    }
  }
  //cuda arrays
  uf4 *norm_d;
  uf4 *pos_d;
  float *surf_d;
  ui4 *ep_d;
  CUDA_SAFE_CALL( hipMalloc((void **) &norm_d,        nbe*sizeof(uf4  )) );
  CUDA_SAFE_CALL( hipMalloc((void **) &pos_d ,(nvert+nbe)*sizeof(uf4  )) );
  CUDA_SAFE_CALL( hipMalloc((void **) &surf_d,        nbe*sizeof(float)) );
  CUDA_SAFE_CALL( hipMalloc((void **) &ep_d  ,        nbe*sizeof(ui4  )) );
  CUDA_SAFE_CALL( hipMemcpy((void *) norm_d,(void *) norma,         nbe*sizeof(uf4  ), hipMemcpyHostToDevice) );
  CUDA_SAFE_CALL( hipMemcpy((void *) pos_d ,(void *) posa , (nvert+nbe)*sizeof(uf4  ), hipMemcpyHostToDevice) );
  CUDA_SAFE_CALL( hipMemcpy((void *) ep_d  ,(void *) ep   ,         nbe*sizeof(ui4  ), hipMemcpyHostToDevice) );
  cout << " [OK]" << endl;
  cout << "\n\tInformation:" << endl;
  cout << "\tOrigin of domain:           \t(" << xmin << ", " << ymin << ", " << zmin << ")\n";
  cout << "\tSize of domain:             \t(" << xmax-xmin << ", " << ymax-ymin << ", " << zmax-zmin << ")\n";
  cout << "\tNumber of vertices:         \t" << nvert << endl;
  cout << "\tNumber of boundary elements:\t" << nbe << "\n\n";

  //calculate surface and position of boundary elements
  cout << "Calculating surface and position of boundary elements ...";
  fflush(stdout);
  int numThreads, numBlocks;
  numThreads = threadsPerBlock;
  numBlocks = (int) ceil((float)nbe/(float)numThreads);
  numBlocks = min(numBlocks,maxblock);
  Lock lock;
  float xminp = 1e10, xminn = 1e10;
  float nminp = 0., nminn = 0.;
  float *xminp_d, *xminn_d;
  float *nminp_d, *nminn_d;
  CUDA_SAFE_CALL( hipMalloc((void **) &xminp_d, sizeof(float)) );
  CUDA_SAFE_CALL( hipMalloc((void **) &xminn_d, sizeof(float)) );
  CUDA_SAFE_CALL( hipMalloc((void **) &nminp_d, sizeof(float)) );
  CUDA_SAFE_CALL( hipMalloc((void **) &nminn_d, sizeof(float)) );
  CUDA_SAFE_CALL( hipMemcpy(xminp_d, &xminp, sizeof(float), hipMemcpyHostToDevice) );
  CUDA_SAFE_CALL( hipMemcpy(xminn_d, &xminn, sizeof(float), hipMemcpyHostToDevice) );
  CUDA_SAFE_CALL( hipMemcpy(nminp_d, &nminp, sizeof(float), hipMemcpyHostToDevice) );
  CUDA_SAFE_CALL( hipMemcpy(nminn_d, &nminn, sizeof(float), hipMemcpyHostToDevice) );

  set_bound_elem<<<numBlocks, numThreads>>> (pos_d, norm_d, surf_d, ep_d, nbe, xminp_d, xminn_d, nminp_d, nminn_d, lock, nvert);

  CUDA_SAFE_CALL( hipMemcpy((void *) posa,(void *) pos_d  , (nvert+nbe)*sizeof(uf4  ), hipMemcpyDeviceToHost) );
  CUDA_SAFE_CALL(  hipMemcpy((void *) surf,(void *) surf_d ,         nbe*sizeof(float), hipMemcpyDeviceToHost) );
  CUDA_SAFE_CALL( hipMemcpy(&xminp, xminp_d, sizeof(float), hipMemcpyDeviceToHost) );
  CUDA_SAFE_CALL( hipMemcpy(&xminn, xminn_d, sizeof(float), hipMemcpyDeviceToHost) );
  CUDA_SAFE_CALL( hipMemcpy(&nminp, nminp_d, sizeof(float), hipMemcpyDeviceToHost) );
  CUDA_SAFE_CALL( hipMemcpy(&nminn, nminn_d, sizeof(float), hipMemcpyDeviceToHost) );
  hipFree (xminp_d);
  hipFree (xminn_d);
  hipFree (nminp_d);
  hipFree (nminn_d);
  //host
  cout << " [OK]" << endl;
  cout << "\n\tNormals information:" << endl;
  cout << "\tPositive (n.(0,0,1)) minimum z: " << xminp << " (" << nminp << ")\n";
  cout << "\tNegative (n.(0,0,1)) minimum z: " << xminn << " (" << nminn << ")\n\n";
  if(fabs(nminp) < 1e-6 && fabs(nminn) < 1e-6 && fabs(xminp-1e10) < 1e-6 && fabs(xminn-1e10) < 1e-6){
    cout << "\t=====================================================" << endl;
    cout << "\t!!! WARNING !!!" << endl;
    cout << "\tCould not read normals properly." << endl;
    cout << "\tMaybe a Blender STL file? Save with ParaView instead." << endl;
    cout << "\t=====================================================\n" << endl;
  }

  if (config.GetBoolean("mesh", "swap_normals", false)) {
    cout << "Swapping normals ...";
    fflush(stdout);

    swap_normals<<<numBlocks, numThreads>>> (norm_d, nbe);

    CUDA_SAFE_CALL( hipMemcpy((void *) norma,(void *) norm_d, nbe*sizeof(uf4), hipMemcpyDeviceToHost) );

    cout << " [OK]" << endl;
  }
  cout << endl;

  //periodicity
  uf4 dmin = {xmin,ymin,zmin,0.};
  uf4 dmax = {xmax,ymax,zmax,0.};
  bool per[3] = {false, false, false};
  int *newlink, *newlink_h;
  uf4 *dmin_d, *dmax_d;
  newlink_h = new int[nvert];
  CUDA_SAFE_CALL( hipMalloc((void **) &newlink, nvert*sizeof(float)) );
  CUDA_SAFE_CALL( hipMalloc((void **) &dmin_d ,       sizeof(uf4  )) );
  CUDA_SAFE_CALL( hipMalloc((void **) &dmax_d ,       sizeof(uf4  )) );
  CUDA_SAFE_CALL( hipMemcpy((void *) dmin_d , (void *) &dmin    ,       sizeof(float4), hipMemcpyHostToDevice) );
  CUDA_SAFE_CALL( hipMemcpy((void *) dmax_d , (void *) &dmax    ,       sizeof(float4), hipMemcpyHostToDevice) );
  for(unsigned int idim=0; idim<3; idim++){
    string pstring = (idim==0) ? "x" : ((idim==1) ? "y" : "z");

    if (config.GetBoolean("periodicity", pstring, false)) {
      per[idim] = true;
      cout << "Updating links for " << pstring << "-periodicity ...";
      fflush(stdout);
      for(unsigned int i=0; i<nvert; i++)
        newlink_h[i] = -1;
      CUDA_SAFE_CALL( hipMemcpy((void *) newlink, (void *) newlink_h, nvert*sizeof(int)   , hipMemcpyHostToDevice) );
      numBlocks = (int) ceil((float)max(nvert,nbe)/(float)numThreads);
      numBlocks = min(numBlocks,maxblock);

      find_links <<<numBlocks, numThreads>>> (pos_d, nvert, dmax_d, dmin_d, dr, newlink, idim);
      periodicity_links<<<numBlocks,numThreads>>>(pos_d, ep_d, nvert, nbe, dmax_d, dmin_d, dr, newlink, idim);

      CUDA_SAFE_CALL( hipMemcpy((void *) posa,(void *) pos_d, (nvert+nbe)*sizeof(uf4), hipMemcpyDeviceToHost) );
      //if(err!=0) return err;
      //host
      cout << " [OK]" << endl;
    }
  }
  CUDA_SAFE_CALL( hipMemcpy((void *) ep  ,(void *) ep_d ,         nbe*sizeof(ui4), hipMemcpyDeviceToHost) );
  CUDA_SAFE_CALL( hipFree(newlink) );
  delete [] newlink_h;

  //calculate volume of vertex particles
  cout << "\nCalculating volume of vertex particles ...";
  fflush(stdout);
  float eps=dr/(float)gres*1e-4;
  int *trisize, *trisize_h;
  float *vol_d;
  bool *per_d;
  trisize_h = new int[nvert];
  for(unsigned int i=0; i<nvert; i++)
    trisize_h[i] = 0;
  CUDA_SAFE_CALL( hipMalloc((void **) &trisize, nvert*sizeof(int  )) );
  CUDA_SAFE_CALL( hipMalloc((void **) &vol_d  , nvert*sizeof(float)) );
  CUDA_SAFE_CALL( hipMalloc((void **) &per_d  ,     3*sizeof(bool )) );
  CUDA_SAFE_CALL( hipMemcpy((void *) per_d  , (void *) per      ,     3*sizeof(bool), hipMemcpyHostToDevice) );
  CUDA_SAFE_CALL( hipMemcpy((void *) trisize, (void *) trisize_h, nvert*sizeof(int) , hipMemcpyHostToDevice) );
  numBlocks = (int) ceil((float)nvert/(float)numThreads);
  numBlocks = min(numBlocks,maxblock);
  delete [] trisize_h;

  calc_trisize <<<numBlocks, numThreads>>> (ep_d, trisize, nbe);
#ifndef bdebug
  calc_vert_volume <<<numBlocks, numThreads>>> (pos_d, norm_d, ep_d, vol_d, trisize, dmin_d, dmax_d, nvert, nbe, dr, eps, per_d);
#else
  uf4 *debug, *debug_d;
  int debugs = pow((gres*2+1),3);
  float *debugp, *debugp_d;
  debugp = new float [100];
  debug = new uf4[debugs];
  CUDA_SAFE_CALL( hipMalloc((void **) &debug_d, debugs*sizeof(uf4)) );
  CUDA_SAFE_CALL( hipMalloc((void **) &debugp_d, 100*sizeof(float)) );

  calc_vert_volume <<<numBlocks, numThreads>>> (pos_d, norm_d, ep_d, vol_d, trisize, dmin_d, dmax_d, nvert, nbe, dr, eps, per_d, debug_d, debugp_d);

  CUDA_SAFE_CALL( hipMemcpy((void*) debug, (void*) debug_d, debugs*sizeof(uf4), hipMemcpyDeviceToHost) );
  CUDA_SAFE_CALL( hipMemcpy((void*) debugp, (void*) debugp_d, 100*sizeof(float), hipMemcpyDeviceToHost) );
  for(int i=0; i<30; i++){
    cout << i << " " << debugp[i] << endl;
  }
#endif

  CUDA_SAFE_CALL( hipMemcpy((void *) vola,(void *) vol_d, nvert*sizeof(float), hipMemcpyDeviceToHost) );
  //cout << endl;
  //for(int i=0; i<30; i++) cout << i << " " << vola[i] << endl;
  hipFree( vol_d   );

  cout << " [OK]" << endl;

  hipFree(vol_d   );

  // seting epsilon to something meaningful based on the geometry size
  eps = 1e-10f;
  for(unsigned int i=0; i<3; i++)
    eps = max((dmax.a[i]-dmin.a[i])*1e-5f,eps);

  // searching for special boundaries
  int *sbid;
  int *sbid_d;
  bool sbpresent = false;
  bool needsUpdate = false;
  bool *needsUpdate_d;
  string cfname;
  int flen = strlen(argv[1]);
  int sbi = 0;
  while(true){
    sbi++;
    cfname = configfname.substr(0,configfname.length()-4);
    cfname += "_sbgrid_";
    stringstream ss;
    ss << sbi;
    cfname += ss.str();
    cfname += ".stl";
    string option = "mesh" + ss.str();
    cfname = config.Get("special_boundary_grids", option, cfname);

    cout << "\nChecking whether special boundary grid #" << sbi << " (" << cfname << ") is available ...";
    stl_file.open(cfname.c_str(), ios::in);
    if(!stl_file.is_open()){
      cout << " [NO]" << endl;
      break;
    }
    else{
      cout << " [YES]" << endl;
      cout << "Checking whether special boundary stl file #" << sbi << " is binary ...";
      bool issolid = true;
      char header[6] = "solid";
      for (int i=0; i<5; i++){
        char dum;
        stl_file.read((char *)&dum, sizeof(char));
        if(dum!=header[i]){
          issolid = false;
          break;
        }
      }
      stl_file.close();
      if(issolid){
        cout << " [NO]" << endl;
        break;
      }
      else{
        cout << " [YES]" << endl;
        // reopen file in binary mode
        stl_file.open(cfname.c_str(), ios::in | ios::binary);
      }
    }
    int sbnvert, sbnbe;
    uf4 *sbposa;
    ui4 *sbep;
    // read header
    for (int i=0; i<20; i++){
      float dum;
      stl_file.read((char *)&dum, sizeof(float));
    }
    // get number of facets
    stl_file.read((char *)&num_of_facets, sizeof(int));
    cout << "Reading " << num_of_facets << " facets of special boundary geometry #" << sbi << " ...";
    fflush(stdout);

    // define variables
    pos.clear();
    epv.clear();
    for(int i=0;i<3;i++){
      ddum[i] = 0.;
      idum[i] = 0;
    }

    // read data
    through = 0;
    while ((through < num_of_facets) & (!stl_file.eof()))
    {
      for (int i=0; i<12; i++)
      {
        stl_file.read((char *)&m_v_floats[i], sizeof(float));
      }
      for(int j=0;j<3;j++){
        for(int i=0;i<3;i++) ddum[i] = (float)m_v_floats[i+3*(j+1)];
        int k = 0;
        bool found = false;
        for(it = pos.begin(); it < pos.end(); it++){
          float diff = 0;
          for(int i=0;i<3;i++) diff += pow((*it)[i]-ddum[i],2);
          diff = sqrt(diff);
          if(diff < 1e-5*dr){
            idum[j] = k;
            found = true;
            break;
          }
          k++;
        }
        if(!found){
          pos.push_back(ddum);
          idum[j] = k;
        }
      }
      epv.push_back(idum);
      stl_file.read((char *)&attribute, sizeof(short));
      through++;
    }
    stl_file.close();
    if(num_of_facets != epv.size()){
      cout << " [FAILED]" << endl;
      return READ_ERROR;
    }
    sbnvert = pos.size();
    sbnbe   = epv.size();
    //create and copy vectors to arrays
    sbposa  = new uf4   [sbnvert];
    sbep    = new ui4   [sbnbe];
    for(unsigned int i=0; i<max(sbnvert,sbnbe); i++){
      if(i<sbnbe){
        for(int j=0; j<3; j++){
          sbep[i].a[j] = epv[i][j];
        }
      }
      if(i<sbnvert){
        for(unsigned int j=0; j<3; j++)
          sbposa[i].a[j] = pos[i][j];
      }
    }
    pos.clear();
    epv.clear();
    sbpresent = true;
    cout << " [OK]" << endl;

    // after reading in data for special boundaries copy data to gpu and identify interior boundary segments and surrounded vertex particles
    sbid = new int[nbe+nvert];
    for(int i=0; i<nbe+nvert; i++) sbid[i] = 0;
    uf4 *sbpos_d;
    ui4 *sbep_d;
    if(sbi==1){
      CUDA_SAFE_CALL( hipMalloc((void **) &needsUpdate_d , sizeof(bool)) );
      CUDA_SAFE_CALL( hipMalloc((void **) &sbid_d , (nvert+nbe)*sizeof(int)) );
      CUDA_SAFE_CALL( hipMemcpy((void *) needsUpdate_d, (void *) &needsUpdate, sizeof(bool), hipMemcpyHostToDevice) );
      CUDA_SAFE_CALL( hipMemcpy((void *) sbid_d, (void *) sbid, (nvert+nbe)*sizeof(int), hipMemcpyHostToDevice) );
    }
    CUDA_SAFE_CALL( hipMalloc((void **) &sbpos_d  ,     sbnvert*sizeof(uf4)) );
    CUDA_SAFE_CALL( hipMalloc((void **) &sbep_d   ,       sbnbe*sizeof(ui4)) );
    CUDA_SAFE_CALL( hipMemcpy((void *) sbpos_d, (void *) sbposa, sbnvert*sizeof(uf4), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy((void *) sbep_d , (void *) sbep  ,   sbnbe*sizeof(ui4), hipMemcpyHostToDevice) );
    numBlocks = (int) ceil((float)nbe/(float)numThreads);
    numBlocks = min(numBlocks,maxblock);

    identifySpecialBoundarySegments<<<numBlocks, numThreads>>> (pos_d, ep_d, nvert, nbe, sbpos_d, sbep_d, sbnbe, eps, sbid_d, sbi);

    numBlocks = (int) ceil((float)nvert/(float)numThreads);
    numBlocks = min(numBlocks,maxblock);

    identifySpecialBoundaryVertices<<<numBlocks, numThreads>>> (sbid_d, sbi, trisize, nvert);

    numBlocks = (int) ceil((float)nbe/(float)numThreads);
    numBlocks = min(numBlocks,maxblock);

    checkForSingularSegments<<<numBlocks, numThreads>>> (pos_d, ep_d, norm_d, surf_d, nvert, nbe, sbid_d, sbi, dr, eps, per_d, dmin_d, dmax_d, needsUpdate_d);

    hipFree( sbpos_d );
    hipFree( sbep_d  );
  }
  hipFree( trisize );
  if(sbpresent){
    CUDA_SAFE_CALL( hipMemcpy((void *) sbid, (void *) sbid_d, (nbe+nvert)*sizeof(int) , hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipMemcpy((void *) &needsUpdate, (void *) needsUpdate_d, sizeof(bool), hipMemcpyDeviceToHost) );
    if (needsUpdate) {
      cout << "\nInformation: Special boundaries required repositioning of some segments" << endl;
      // copy ep, surf back to host
      CUDA_SAFE_CALL( hipMemcpy((void *) ep  ,(void *) ep_d ,         nbe*sizeof(ui4), hipMemcpyDeviceToHost) );
      CUDA_SAFE_CALL(  hipMemcpy((void *) surf,(void *) surf_d ,         nbe*sizeof(float), hipMemcpyDeviceToHost) );
      CUDA_SAFE_CALL( hipMemcpy((void *) posa,(void *) pos_d, (nvert+nbe)*sizeof(uf4), hipMemcpyDeviceToHost) );
    }
    hipFree( sbid_d  );
  }

  hipFree(surf_d  );

  //setting up fluid particles
  cout << "\nDefining fluid particles ..." << endl;

  cfname = configfname.substr(0,configfname.length()-4);
  cfname += "_fshape.stl";
  cfname = config.Get("mesh", "fshape", cfname);

  cout << "Checking whether fluid geometry (" << cfname << ") is available ...";
  fflush(stdout);

  ifstream fstl_file (cfname.c_str(), ios::in);
  if(!fstl_file.is_open()){
    cout << " [NO]" << endl;
  }
  else{
    cout << " [YES]" << endl;
    cout << "Checking whether fluid geometry stl file is binary ...";
    fflush(stdout);
    bool issolid = true;
    char header[6] = "solid";
    for (int i=0; i<5; i++){
      char dum;
      fstl_file.read((char *)&dum, sizeof(char));
      if(dum!=header[i]){
        issolid = false;
        break;
      }
    }
    fstl_file.close();
    if(issolid){
      cout << " [NO]" << endl;
    }
    else{
      cout << " [YES]" << endl;
      // reopen file in binary mode
      fstl_file.open(cfname.c_str(), ios::in | ios::binary);
      if(!fstl_file.is_open()){
        cout << "Error: could not reopen fluid geometry file in binary mode" << endl;
        return -1;
      }
    }
  }

  bool set = true;
  bool firstfgeom = true;
  unsigned int cnvert, cnbe;
  uf4 *cnorma, *cposa;
  ui4 *cep;
  unsigned int nfluid = 0;
  unsigned int maxf = 0, maxfn;
  int opt;
  unsigned int *fpos, *fpos_d;
  unsigned int *nfi_d;

  set = config.GetBoolean("fluid_container", "use", false);

  if(set){
    // From here on dmin, dmax represent the fluid container and no longer the domain container.
    dmin.a[0] = config.GetReal("fluid_container", "xmin", 1e9);
    dmin.a[1] = config.GetReal("fluid_container", "ymin", 1e9);
    dmin.a[2] = config.GetReal("fluid_container", "zmin", 1e9);
    dmax.a[0] = config.GetReal("fluid_container", "xmax", -1e9);
    dmax.a[1] = config.GetReal("fluid_container", "ymax", -1e9);
    dmax.a[2] = config.GetReal("fluid_container", "zmax", -1e9);
    cout << "Fluid container specified:" << endl;
    cout << "Min coordinates (" << dmin.a[0] << ", " << dmin.a[1] << ", " << dmin.a[2] << ")" << endl;
    cout << "Max coordinates (" << dmax.a[0] << ", " << dmax.a[1] << ", " << dmax.a[2] << ")" << endl;
    CUDA_SAFE_CALL( hipMemcpy((void *) dmin_d , (void *) &dmin    ,       sizeof(float4), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy((void *) dmax_d , (void *) &dmax    ,       sizeof(float4), hipMemcpyHostToDevice) );
  }
  else{
    cout << "Using whole geometry as fluid container." << endl;
  }

  maxfn = (int)floor((dmax.a[0]-dmin.a[0]+eps)/dr+1)*floor((dmax.a[1]-dmin.a[1]+eps)/dr+1)*floor((dmax.a[2]-dmin.a[2]+eps)/dr+1);
  maxf = (int)ceil(float(maxfn)/8./((float)sizeof(unsigned int)));
  fpos = new unsigned int [maxf];
  CUDA_SAFE_CALL( hipMalloc((void **) &fpos_d, maxf*sizeof(unsigned int)) );
  CUDA_SAFE_CALL( hipMalloc((void **) &nfi_d, sizeof(unsigned int)) );
  for(unsigned int i=0; i<maxf; i++) fpos[i] = 0;
  CUDA_SAFE_CALL( hipMemcpy((void *) fpos_d, (void *) fpos, maxf*sizeof(unsigned int), hipMemcpyHostToDevice) );

  bool continueFill = true;
  int nFill = 0;
  while(continueFill){
    stringstream fillSection;
    fillSection << "fill_" << nFill;
    string option=config.Get(fillSection.str(), "option", "box");
    if (option=="geometry")
      opt = 2;
    else
      opt = 1;
    cout << "\nOption for fill #" << nFill << ": " << option << endl;
    xmin = xmax = ymin = ymax = zmin = zmax = 0.;

    // data for geometry bounding grid and fluid bounding grid
    unsigned int fnvert=0, fnbe=0;
    uf4 *fposa=NULL, *fnorma=NULL;
    ui4 *fep=NULL;

    if(opt==1){ // fluid based on rectangular box
      xmin = config.GetReal(fillSection.str(), "xmin", 1e9);
      ymin = config.GetReal(fillSection.str(), "ymin", 1e9);
      zmin = config.GetReal(fillSection.str(), "zmin", 1e9);
      xmax = config.GetReal(fillSection.str(), "xmax", -1e9);
      ymax = config.GetReal(fillSection.str(), "ymax", -1e9);
      zmax = config.GetReal(fillSection.str(), "zmax", -1e9);
      cout << "Fluid box specified:" << endl;
      cout << "Min coordinates (" << xmin << ", " << ymin << ", " << zmin << ")" << endl;
      cout << "Max coordinates (" << xmax << ", " << ymax << ", " << zmax << ")" << endl;
      if(xmax-xmin<1e-5*dr || ymax-ymin<1e-5*dr || zmax-zmin<1e-5*dr){
        cout << "\nMistake in input for fluid box dimensions" << endl;
        cout << "Fluid particle definition ... [FAILED]" << endl;
        return FLUID_NDEF;
      }
      numBlocks = (int) ceil((float)maxf/(float)numThreads);
      numBlocks = min(numBlocks,maxblock);

      Lock lock_f;
      unsigned int nfi=0;
      CUDA_SAFE_CALL( hipMemcpy((void *) nfi_d, (void *) &nfi, sizeof(unsigned int), hipMemcpyHostToDevice) );

      fill_fluid<<<numBlocks, numThreads>>> (fpos_d, nfi_d, xmin, xmax, ymin, ymax, zmin, zmax, dmin_d, dmax_d, eps, dr, lock_f);

      CUDA_SAFE_CALL( hipMemcpy((void *) &nfi, (void *) nfi_d, sizeof(unsigned int), hipMemcpyDeviceToHost) );
      nfluid += nfi;

    }

    else if(opt==2){ // fluid based on geometry
      // get seed point
      float spos[3], dr_wall;
      spos[0] = config.GetReal(fillSection.str(), "xseed", 1e9);
      spos[1] = config.GetReal(fillSection.str(), "yseed", 1e9);
      spos[2] = config.GetReal(fillSection.str(), "zseed", 1e9);
      cout << "Seed point (" << spos[0] << ", " << spos[1] << ", " << spos[2] << ")" << endl;
      dr_wall = config.GetReal(fillSection.str(), "dr_wall", dr);
      cout << "Distance from fluid particles to vertices and segments: " << dr_wall << endl;
      // initialize placing of seed point
      int ispos = (int)round((spos[0]-dmin.a[0]+eps)/dr);
      int jspos = (int)round((spos[1]-dmin.a[1]+eps)/dr);
      int kspos = (int)round((spos[2]-dmin.a[2]+eps)/dr);
      int idimg = (int)floor((dmax.a[0]-dmin.a[0]+eps)/dr+1);
      int jdimg = (int)floor((dmax.a[1]-dmin.a[1]+eps)/dr+1);
      int sInd = ispos + jspos*idimg + kspos*idimg*jdimg;

      // initialize geometry if first run
      if(firstfgeom){
        firstfgeom = false;

        hipFree(norm_d  );
        hipFree(pos_d   );
        hipFree(ep_d    );

        // copy stl geometry to f* arrays
        fnvert = nvert;
        fnbe = nbe;
        fep = new ui4 [fnbe];
        fnorma = new uf4 [fnbe];
        fposa = new uf4 [fnvert];
        unsigned int inbe = 0;
        for(unsigned int i=0; i<max(fnvert,fnbe); i++){
          if(i<fnbe){
            // if a fluid container was set then remove all normals and ep of segments that are outside the box + 2dr
            if(! set ||
               (fabs(posa[i+nvert].a[0] - (dmax.a[0]+dmin.a[0])/2.0f) < (dmax.a[0]-dmin.a[0])/2.0f + 2.0f*dr &&
                fabs(posa[i+nvert].a[1] - (dmax.a[1]+dmin.a[1])/2.0f) < (dmax.a[1]-dmin.a[1])/2.0f + 2.0f*dr &&
                fabs(posa[i+nvert].a[2] - (dmax.a[2]+dmin.a[2])/2.0f) < (dmax.a[2]-dmin.a[2])/2.0f + 2.0f*dr   )){
              fep[inbe] = ep[i];
              fnorma[inbe] = norma[i];
              inbe++;
            }
          }
          // all vertices will be copied regardless of their location
          if(i<fnvert)
            fposa[i] = posa[i];
        }
        if(set)
          fnbe = inbe;

        // read fluid geometry
        // read header
        for (int i=0; i<20; i++){
        float dum;
        fstl_file.read((char *)&dum, sizeof(float));
        }
        // get number of facets
        fstl_file.read((char *)&num_of_facets, sizeof(int));
        cout << "Reading " << num_of_facets << " facets of fluid geometry ...";
        fflush(stdout);

        // define variables
        pos.clear();
        norm.clear();
        epv.clear();
        for(int i=0;i<3;i++){
          ddum[i] = 0.;
          idum[i] = 0;
        }

        // read data
        through = 0;
        while ((through < num_of_facets) & (!fstl_file.eof()))
        {
          for (int i=0; i<12; i++){
            fstl_file.read((char *)&m_v_floats[i], sizeof(float));
          }
          for(int j=0;j<3;j++){
            for(int i=0;i<3;i++) ddum[i] = (float)m_v_floats[i+3*(j+1)];
            int k = 0;
            bool found = false;
            for(it = pos.begin(); it < pos.end(); it++){
              float diff = 0;
              for(int i=0;i<3;i++) diff += pow((*it)[i]-ddum[i],2);
              diff = sqrt(diff);
              if(diff < 1e-5*dr){
                idum[j] = k+fnvert;
                found = true;
                break;
              }
              k++;
            }
            if(!found){
              pos.push_back(ddum);
              idum[j] = k+fnvert;
            }
          }
          // get normal of triangle
          float lenNorm = 0.0;
          for(int i=0;i<3;i++){
            ddum[i] = (float)m_v_floats[i];
            lenNorm += ddum[i]*ddum[i];
          }
          // this is for blender if stl files are saved without normals
          // here we don't care for the orientation so let's just compute it
          if(lenNorm < eps){
            uf4 v10, v20;
            for(int i=0; i<3; i++){
              v10.a[i] = pos[idum[1]-fnvert][i] - pos[idum[0]-fnvert][i];
              v20.a[i] = pos[idum[2]-fnvert][i] - pos[idum[0]-fnvert][i];
            }
            uf4 tnorm = cross(v10, v20);
            for(int i=0; i<3; i++)
              ddum[i] = tnorm.a[i];
          }
          norm.push_back(ddum);
          epv.push_back(idum);
          fstl_file.read((char *)&attribute, sizeof(short));
          through++;
        }
        fstl_file.close();
        if(num_of_facets != norm.size()){
          cout << " [FAILED]" << endl;
          return READ_ERROR;
        }
        cnvert = pos.size();
        cnbe   = norm.size();
        cout << " [OK]" << endl;
        cout << "Merging arrays and preparing device for filling ...";
        fflush(stdout);
        //create and copy vectors to arrays
        cnorma = new uf4   [fnbe];
        cposa  = new uf4   [fnvert];
        cep    = new ui4   [fnbe];
        for(unsigned int i=0; i<max(fnbe,fnvert); i++){
          if(i<fnbe){
            cnorma[i] = fnorma[i];
            cep   [i] = fep   [i];
          }
          if(i<fnvert){
            cposa [i] = fposa [i];
          }
        }
        delete [] fnorma;
        delete [] fposa;
        delete [] fep;
        fnorma = new uf4   [fnbe+cnbe];
        fposa  = new uf4   [fnvert+cnvert];
        fep    = new ui4   [fnbe+cnbe];
        for(unsigned int i=0; i<max(fnbe,fnvert); i++){
          if(i<fnbe){
            fnorma[i] = cnorma[i];
            fep   [i] = cep   [i];
          }
          if(i<fnvert){
            fposa [i] = cposa [i];
          }
        }
        delete [] cnorma;
        delete [] cposa;
        delete [] cep;
        for(unsigned int i=0; i<max(cnvert,cnbe); i++){
          if(i<cnbe){
            for(int j=0; j<3; j++){
              fnorma[i+fnbe].a[j] = norm[i][j];
              fep[i+fnbe].a[j] = epv[i][j];
            }
          }
          if(i<cnvert){
            for(int j=0; j<3; j++)
              fposa[i+fnvert].a[j] = pos[i][j];
          }
        }
        fnvert += cnvert;
        fnbe += cnbe;
        pos.clear();
        epv.clear();
        norm.clear();
        CUDA_SAFE_CALL( hipMalloc((void **) &norm_d,   fnbe*sizeof(uf4  )) );
        CUDA_SAFE_CALL( hipMalloc((void **) &pos_d , fnvert*sizeof(uf4  )) );
        CUDA_SAFE_CALL( hipMalloc((void **) &ep_d  ,   fnbe*sizeof(ui4  )) );
        CUDA_SAFE_CALL( hipMemcpy((void *) norm_d, (void *) fnorma,   fnbe*sizeof(uf4), hipMemcpyHostToDevice) );
        CUDA_SAFE_CALL( hipMemcpy((void *) pos_d , (void *) fposa , fnvert*sizeof(uf4), hipMemcpyHostToDevice) );
        CUDA_SAFE_CALL( hipMemcpy((void *) ep_d  , (void *) fep   ,   fnbe*sizeof(ui4), hipMemcpyHostToDevice) );

        numBlocks = (int) ceil((float)maxf/(float)numThreads);
        numBlocks = min(numBlocks,maxblock);
        cout << " [OK]" << endl;
      } // end firstfgeom

      unsigned int nfi;
      unsigned int iteration = 0;
      do{
        Lock lock_f;
        iteration++;
        nfi = 0;
        CUDA_SAFE_CALL( hipMemcpy((void *) nfi_d, (void *) &nfi, sizeof(unsigned int), hipMemcpyHostToDevice) );

        fill_fluid_complex<<<numBlocks, numThreads>>> (fpos_d, nfi_d, norm_d, ep_d, pos_d, fnbe, dmin_d, dmax_d, eps, dr, sInd, lock_f, cnbe, dr_wall, iteration);

        CUDA_SAFE_CALL( hipMemcpy((void *) &nfi, (void *) nfi_d, sizeof(unsigned int), hipMemcpyDeviceToHost) );
        nfluid += nfi;
      } while(nfi > 0 && iteration < max_iterations);
    }

    stringstream fillSectionTest;
    fillSectionTest << "fill_" << (nFill+1);
    if (config.Get(fillSectionTest.str(), "option", "UNKNOWN") == "UNKNOWN") {
      continueFill = false;
      if (!firstfgeom) {
        delete [] fposa;
        delete [] fnorma;
        delete [] fep;
      }
    }
    nFill++;
  }
  CUDA_SAFE_CALL( hipMemcpy((void *) fpos, (void *) fpos_d, maxf*sizeof(unsigned int), hipMemcpyDeviceToHost) );
  cout << "\nCreation of " << nfluid << " fluid particles completed. [OK]" << endl;
  hipFree( fpos_d );
  hipFree( nfi_d  );
  hipFree( norm_d );
  hipFree( pos_d  );
  hipFree( ep_d   );

  //prepare output structure for particles
  cout << "Creating and initializing of output buffer of particles ...";
  fflush(stdout);
  OutBuf *buf, *beBuf;
#ifndef bdebug
  unsigned int nelem = nvert+nbe+nfluid;
#else
  unsigned int nelem = nvert+nbe+nfluid+debugs;
#endif
  buf = new OutBuf[nelem];
  // buffer for boundary elementss
  beBuf = new OutBuf[nbe];
  int k=0;
  unsigned int m,n,imin[3];
  float fluid_vol = pow(dr,3);
  imin[0] = int(floor((dmax.a[0]-dmin.a[0]+eps)/dr))+1;
  imin[1] = int(floor((dmax.a[1]-dmin.a[1]+eps)/dr))+1;
  imin[2] = int(floor((dmax.a[2]-dmin.a[2]+eps)/dr))+1;
  //free particles
  for(unsigned int j=0; j<maxfn; j++){
    int i = j/bitPerUint;
    int l = j%bitPerUint;
    m = 1 << l;
    if(fpos[i] & m){
      m = j/(imin[1]*imin[0]);
      buf[k].z = dmin.a[2]+dr*(float)m;
      n = j%(imin[1]*imin[0]);
      m = n/imin[0];
      buf[k].y = dmin.a[1]+dr*(float)m;
      m = n%imin[0];
      buf[k].x = dmin.a[0]+dr*(float)m;
      buf[k].nx = 0.;
      buf[k].ny = 0.;
      buf[k].nz = 0.;
      buf[k].vol = fluid_vol;
      buf[k].surf = 0.;
      buf[k].kpar = 1;
      buf[k].kfluid = 1;
      buf[k].kent = 0;
      buf[k].kparmob = 0;
      buf[k].iref = k;
      buf[k].ep1 = 0;
      buf[k].ep2 = 0;
      buf[k].ep3 = 0;
      k++;
    }
  }
  //vertex particles
  int *nvshift;
  nvshift = new int[nvert];
  for(unsigned int i=0; i<nvert; i++)
    nvshift[i] = 0;
  int ishift = 0;
  for(unsigned int i=0; i<nvert; i++){
    if(posa[i].a[0] < -1e9){
      nelem--;
      ishift++;
      continue;
    }
    nvshift[i] = ishift;
    buf[k].x = posa[i].a[0];
    buf[k].y = posa[i].a[1];
    buf[k].z = posa[i].a[2];
    buf[k].nx = 0.;
    buf[k].ny = 0.;
    buf[k].nz = 0.;
    buf[k].vol = vola[i];
    buf[k].surf = 0.;
    buf[k].kpar = 2;
    buf[k].kfluid = 1;
    if(sbpresent)
      buf[k].kent = sbid[i];
    else
      buf[k].kent = 0;
    buf[k].kparmob = 0;
    buf[k].iref = k;
    buf[k].ep1 = 0;
    buf[k].ep2 = 0;
    buf[k].ep3 = 0;
    k++;
  }
  const unsigned int nCur = k;
  //boundary segments
  //these are preliminarily written into beBuf because we might need to rearrange them
  //count the numbers of special boundary elements
  unsigned int *nsbe, *isbe;
  nsbe = new unsigned int[sbi];
  isbe = new unsigned int[sbi];
  for(unsigned int i=0; i<sbi; i++)
    nsbe[i] = 0;
  for(unsigned int i=nvert; i<nvert+nbe; i++){
    beBuf[k-nCur].x = posa[i].a[0];
    beBuf[k-nCur].y = posa[i].a[1];
    beBuf[k-nCur].z = posa[i].a[2];
    beBuf[k-nCur].nx = norma[i-nvert].a[0];
    beBuf[k-nCur].ny = norma[i-nvert].a[1];
    beBuf[k-nCur].nz = norma[i-nvert].a[2];
    beBuf[k-nCur].vol = 0.;
    beBuf[k-nCur].surf = surf[i-nvert];
    beBuf[k-nCur].kpar = 3;
    beBuf[k-nCur].kfluid = 1;
    if(sbpresent){
      beBuf[k-nCur].kent = sbid[i];
      nsbe[sbid[i]]++;
    }
    else
      beBuf[k-nCur].kent = 0;
    beBuf[k-nCur].kparmob = 0;
    beBuf[k-nCur].iref = k;
    beBuf[k-nCur].ep1 = nfluid+ep[i-nvert].a[0] - nvshift[ep[i-nvert].a[0]];
    beBuf[k-nCur].ep2 = nfluid+ep[i-nvert].a[1] - nvshift[ep[i-nvert].a[1]];
    beBuf[k-nCur].ep3 = nfluid+ep[i-nvert].a[2] - nvshift[ep[i-nvert].a[2]];
    k++;
  }
  // isbe contains the current index of each sbi
  isbe[0] = 0;
  for(unsigned int i=1; i<sbi; i++)
    isbe[i] = nsbe[i-1] + isbe[i-1];
  // copy beBuf into buf while reordering if required
  for(unsigned int i=0; i<nbe; i++){
    unsigned int l = nCur + isbe[beBuf[i].kent];
    buf[l] = beBuf[i];
    buf[l].iref = l;
    isbe[beBuf[i].kent]++;
  }
  delete [] nvshift;
#ifdef bdebug
  //debug
  for(unsigned int i=0; i<debugs; i++){
    buf[k].x = debug[i].a[0];
    buf[k].y = debug[i].a[1];
    buf[k].z = debug[i].a[2];
    buf[k].nx = 0;
    buf[k].ny = 0;
    buf[k].nz = 0;
    buf[k].vol = debug[i].a[3];
    buf[k].surf = 0.;
    buf[k].kpar = 4;
    buf[k].kfluid = 1;
    buf[k].kent = 1;
    buf[k].kparmob = 0;
    buf[k].iref = k;
    buf[k].ep1 = 0;
    buf[k].ep2 = 0;
    buf[k].ep3 = 0;
    k++;
  }
#endif
  cout << " [OK]" << endl;

  //Output of particles
  int err = 0;
  string outfformat = config.Get("output", "format", "vtu");
  if (outfformat == "h5sph")
    opt = 2;
  else
    opt = 1;
  cout << "Output format: " << outfformat << endl;
  string outname = configfname.substr(0,configfname.length()-4);
  outname = config.Get("output", "name", outname);
  if(opt==2){
    outname = "0." + outname + ".h5sph";
    cout << "Writing output to file " << outname << " ...";
    fflush(stdout);
    err = hdf5_output( buf, nelem, outname.c_str());
  }
  else if(opt==1){
    outname += ".vtu";
    cout << "Writing output to file " << outname << " ...";
    fflush(stdout);
    err = vtk_output( buf, nelem, outname.c_str());
  }
  if(err==0){ cout << " [OK]" << endl; }
  else {
    cout << " [FAILED]" << endl;
    return WRITE_FAIL;
  }

  //Free memory
  //Arrays
  delete [] norma;
  delete [] posa;
  delete [] vola;
  delete [] surf;
  delete [] ep;
  delete [] buf;
  delete [] fpos;
  //Cuda
  hipFree( per_d   );
  hipFree( dmin_d  );
  hipFree( dmax_d  );

  //End
  return 0;
}
#endif
