#include "hip/hip_runtime.h"
/***********************************\
 *
 * TODO LIST:
 * - Version 0.5:
 *   - filling via bit field
 *   - filling of complex geometries
 *   - specification of fluid container
 * 	 - check if fluid particles are closer than dr to the wall
 * - Version 0.6:
 *   - in/outflow option
 *   - replace uf4 by uf3 (class with float[3])
 *   - while doing calculations on kernel check
 *     files on host and maybe read them already
 *   - clean up code (free norm, ep when they
 *     are no longer needed and not at the end)
 * - Version 0.7:
 *   - CSG for volume computation
 *
\***********************************/

#ifndef CRIXUS_CU
#define CRIXUS_CU

#include <iostream>
#include <fstream>
#include <vector>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "cuda_local.cuh"
#include "crixus.h"
#include "return.h"
#include "crixus_d.cuh"
#include "lock.cuh"

using namespace std;

int crixus_main(int argc, char** argv){
	//host
	cout << endl;
	cout << "\t*********************************" << endl;
	cout << "\t*                               *" << endl;
	cout << "\t*          C R I X U S          *" << endl;
	cout << "\t*                               *" << endl;
	cout << "\t*********************************" << endl;
	cout << "\t* Version: 0.5                  *" << endl;
	cout << "\t* Date   : 04.12.2013           *" << endl;
	cout << "\t* Authors: Arno Mayrhofer       *" << endl;
	cout << "\t*          Christophe Kassiotis *" << endl;
	cout << "\t*          F-X Morel            *" << endl;
	cout << "\t*          Martin Ferrand       *" << endl;
	cout << "\t*          Agnes Leroy          *" << endl;
	cout << "\t*          Antoine Joly         *" << endl;
	cout << "\t*********************************" << endl;
	cout << endl;
	float m_v_floats[12];
	unsigned int through;
	short attribute;
	unsigned int num_of_facets;
  const unsigned int bitPerUint = 8*sizeof(unsigned int);

  if(argc==1){
		cout << "No file specified." << endl;
		cout << "Correct use: crixus filename dr" << endl;
		cout << "Example use: crixus box.stl 0.1" << endl;
		return NO_FILE;
	}
	else if(argc==2){
		cout << "No particle discretization specified." << endl;
		cout << "Correct use: crixus filename dr" << endl;
		cout << "Example use: crixus box.stl 0.1" << endl;
		return NO_DR;
	}
	
	//looking for cuda devices without timeout
	cout << "Selecting GPU ...";
	int dcount, maxblock=0, maxthread;
	maxthread = threadsPerBlock;
	bool found = false;
	CUDA_SAFE_CALL( hipGetDeviceCount(&dcount) );
	for (int i=0; i<dcount; i++){
		hipDeviceProp_t prop;
		CUDA_SAFE_CALL( hipGetDeviceProperties(&prop,i) );
		if(!prop.kernelExecTimeoutEnabled){
			found = true;
			CUDA_SAFE_CALL( hipSetDevice(i) );
			maxthread = prop.maxThreadsPerBlock;
			maxblock  = prop.maxGridSize[0];
			cout << " Id: " << i << " (" << maxthread << ", " << maxblock << ") ...";
			if(maxthread < threadsPerBlock){
				cout << " [FAILED]" << endl;
				return MAXTHREAD_TOO_BIG;
			}
			cout << " [OK]" << endl;
			break;
		}
	}
	if(!found){
		hipDeviceProp_t prop;
		CUDA_SAFE_CALL( hipGetDeviceProperties(&prop,0) );
		CUDA_SAFE_CALL( hipSetDevice(0) );
		maxthread = prop.maxThreadsPerBlock;
		maxblock  = prop.maxGridSize[0];
		cout << " Id: " << 0 << " (" << maxthread << ", " << maxblock << ") ...";
		if(maxthread < threadsPerBlock){
			cout << " [FAILED]" << endl;
			return MAXTHREAD_TOO_BIG;
		}
		cout << " [OK]" << endl;
		cout << "\n\tWARNING:" << endl;
		cout << "\tCould not find GPU without timeout." << endl;
		cout << "\tIf execution terminates with timeout reduce gres.\n" << endl;
	}
	if(maxthread != threadsPerBlock){
		cout << "\n\tINFORMATION:" << endl;
		cout << "\tthreadsPerBlock is not equal to maximum number of available threads.\n" << endl;
	}

	//Reading file
	cout << "Opening file " << argv[1] << " ...";
	ifstream stl_file (argv[1], ios::in);
	if(!stl_file.is_open()){
		cout << " [FAILED]" << endl;
	  return FILE_NOT_OPEN;
	}
	cout << " [OK]" << endl;

	cout << "Checking whether stl file is not ASCII ...";
	bool issolid = true;
	char header[6] = "solid";
	for (int i=0; i<5; i++){
		char dum;
		stl_file.read((char *)&dum, sizeof(char));
		if(dum!=header[i]){
			issolid = false;
			break;
		}
	}
	if(issolid){
		cout << " [FAILED]" << endl;
		stl_file.close();
		return STL_NOT_BINARY;
	}
	stl_file.close();
	cout << " [OK]" << endl;

	// reopen file in binary mode
	stl_file.open(argv[1], ios::in | ios::binary);

	// read header
	for (int i=0; i<20; i++){
		float dum;
		stl_file.read((char *)&dum, sizeof(float));
	}
	// get number of facets
	stl_file.read((char *)&num_of_facets, sizeof(int));
	cout << "Reading " << num_of_facets << " facets ...";

	float dr = strtod(argv[2],NULL);
	// define variables
	vector< vector<float> > pos;
	vector< vector<float> > norm;
	vector< vector<float> >::iterator it;
	vector< vector<unsigned int> > epv;
	unsigned int nvert, nbe;
	vector<unsigned int> idum;
	vector<float> ddum;
	for(int i=0;i<3;i++){
		ddum.push_back(0.);
		idum.push_back(0);
	}

	// read data
	through = 0;
	float xmin = 1e10, xmax = -1e10;
	float ymin = 1e10, ymax = -1e10;
	float zmin = 1e10, zmax = -1e10;
	while ((through < num_of_facets) & (!stl_file.eof()))
	{
		for (int i=0; i<12; i++)
		{
			stl_file.read((char *)&m_v_floats[i], sizeof(float));
		}
		for(int i=0;i<3;i++) ddum[i] = (float)m_v_floats[i];
		norm.push_back(ddum);
		for(int j=0;j<3;j++){
			for(int i=0;i<3;i++) ddum[i] = (float)m_v_floats[i+3*(j+1)];
			int k = 0;
			bool found = false;
			for(it = pos.begin(); it < pos.end(); it++){
				float diff = 0;
				for(int i=0;i<3;i++) diff += pow((*it)[i]-ddum[i],2);
				diff = sqrt(diff);
				if(diff < 1e-5*dr){
					idum[j] = k;
					found = true;
					break;
				}
				k++;
			}
			if(!found){
				pos.push_back(ddum);
				xmin = (xmin > ddum[0]) ? ddum[0] : xmin;
				xmax = (xmax < ddum[0]) ? ddum[0] : xmax;
				ymin = (ymin > ddum[1]) ? ddum[1] : ymin;
				ymax = (ymax < ddum[1]) ? ddum[1] : ymax;
				zmin = (zmin > ddum[2]) ? ddum[2] : zmin;
				zmax = (zmax < ddum[2]) ? ddum[2] : zmax;
				idum[j] = k;
			}
		}
		epv.push_back(idum);
		stl_file.read((char *)&attribute, sizeof(short));
		through++;
	}
	stl_file.close();
	if(num_of_facets != norm.size()){
		cout << " [FAILED]" << endl;
		return READ_ERROR;
	}
	nvert = pos.size();
	nbe   = norm.size();
	//create and copy vectors to arrays
	uf4 *norma, *posa;
	float *vola, *surf;
	ui4 *ep;
	norma = new uf4   [nbe];
	posa  = new uf4   [nvert+nbe];
	vola  = new float [nvert];
	surf  = new float [nbe]; //AM-TODO: could go to norma[3]
	ep    = new ui4   [nbe];
	for(unsigned int i=0; i<max(nvert,nbe); i++){
		if(i<nbe){
      for(int j=0; j<3; j++){
		  	norma[i].a[j] = norm[i][j];
		  	ep[i].a[j] = epv[i][j];
      }
		}
		if(i<nvert){
      for(int j=0; j<3; j++)
			  posa[i].a[j] = pos[i][j];
			vola[i] = 0.;
		}
	}
	//cuda arrays
	uf4 *norm_d;
	uf4 *pos_d;
	float *surf_d;
	ui4 *ep_d;
	CUDA_SAFE_CALL( hipMalloc((void **) &norm_d,        nbe*sizeof(uf4  )) );
	CUDA_SAFE_CALL( hipMalloc((void **) &pos_d ,(nvert+nbe)*sizeof(uf4  )) );
	CUDA_SAFE_CALL( hipMalloc((void **) &surf_d,        nbe*sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc((void **) &ep_d  ,        nbe*sizeof(ui4  )) );
	CUDA_SAFE_CALL( hipMemcpy((void *) norm_d,(void *) norma,         nbe*sizeof(uf4  ), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy((void *) pos_d ,(void *) posa , (nvert+nbe)*sizeof(uf4  ), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy((void *) surf_d,(void *) surf ,         nbe*sizeof(float), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy((void *) ep_d  ,(void *) ep   ,         nbe*sizeof(ui4  ), hipMemcpyHostToDevice) );
	cout << " [OK]" << endl;
	cout << "\n\tInformation:" << endl;
	cout << "\tOrigin of domain:           \t(" << xmin << ", " << ymin << ", " << zmin << ")\n";
	cout << "\tSize of domain:             \t(" << xmax-xmin << ", " << ymax-ymin << ", " << zmax-zmin << ")\n";
	cout << "\tNumber of vertices:         \t" << nvert << endl;
	cout << "\tNumber of boundary elements:\t" << nbe << "\n\n";

	//calculate surface and position of boundary elements
	cout << "Calculating surface and position of boundary elements ...";
	int numThreads, numBlocks;
	numThreads = threadsPerBlock;
	numBlocks = (int) ceil((float)nbe/(float)numThreads);
	numBlocks = min(numBlocks,maxblock);
	Lock lock;
	float xminp = 1e10, xminn = 1e10;
	float nminp = 0., nminn = 0.;
	float *xminp_d, *xminn_d;
	float *nminp_d, *nminn_d;
	CUDA_SAFE_CALL( hipMalloc((void **) &xminp_d, sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc((void **) &xminn_d, sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc((void **) &nminp_d, sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc((void **) &nminn_d, sizeof(float)) );
	CUDA_SAFE_CALL( hipMemcpy(xminp_d, &xminp, sizeof(float), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(xminn_d, &xminn, sizeof(float), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(nminp_d, &nminp, sizeof(float), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(nminn_d, &nminn, sizeof(float), hipMemcpyHostToDevice) );

	set_bound_elem<<<numBlocks, numThreads>>> (pos_d, norm_d, surf_d, ep_d, nbe, xminp_d, xminn_d, nminp_d, nminn_d, lock, nvert);

	CUDA_SAFE_CALL( hipMemcpy((void *) posa,(void *) pos_d  , (nvert+nbe)*sizeof(uf4  ), hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL(	hipMemcpy((void *) surf,(void *) surf_d ,         nbe*sizeof(float), hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy(&xminp, xminp_d, sizeof(float), hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy(&xminn, xminn_d, sizeof(float), hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy(&nminp, nminp_d, sizeof(float), hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy(&nminn, nminn_d, sizeof(float), hipMemcpyDeviceToHost) );
	hipFree (xminp_d);
	hipFree (xminn_d);
	hipFree (nminp_d);
	hipFree (nminn_d);
	//host
	cout << " [OK]" << endl;
	cout << "\n\tNormals information:" << endl;
	cout << "\tPositive (n.(0,0,1)) minimum z: " << xminp << " (" << nminp << ")\n";
	cout << "\tNegative (n.(0,0,1)) minimum z: " << xminn << " (" << nminn << ")\n\n";
  if(fabs(nminp) < 1e-6 && fabs(nminn) < 1e-6 && fabs(xminp-1e10) < 1e-6 && fabs(xminn-1e10) < 1e-6){
    cout << "\t=====================================================" << endl;
    cout << "\t!!! WARNING !!!" << endl;
    cout << "\tCould not read normals properly." << endl;
    cout << "\tMaybe a Blender STL file? Save with ParaView instead." << endl;
    cout << "\t=====================================================\n" << endl;
  }
	char cont= 'n';
	do{
		if(cont!='n') cout << "Wrong input. Answer with y or n." << endl;
		cout << "Swap normals (y/n): ";
		cin >> cont;
	}while(cont!='y' && cont!='n');
	if(cont=='y'){
		cout << "Swapping normals ...";
		
		swap_normals<<<numBlocks, numThreads>>> (norm_d, nbe);

	  CUDA_SAFE_CALL( hipMemcpy((void *) norma,(void *) norm_d, nbe*sizeof(uf4), hipMemcpyDeviceToHost) );

		cout << " [OK]" << endl;
	}
	cout << endl;

  //periodicity
	uf4 dmin = {xmin,ymin,zmin,0.};
	uf4 dmax = {xmax,ymax,zmax,0.};
	bool per[3] = {false, false, false};
	int *newlink, *newlink_h;
	uf4 *dmin_d, *dmax_d;
	newlink_h = new int[nvert];
	CUDA_SAFE_CALL( hipMalloc((void **) &newlink, nvert*sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc((void **) &dmin_d ,       sizeof(uf4  )) );
	CUDA_SAFE_CALL( hipMalloc((void **) &dmax_d ,       sizeof(uf4  )) );
	CUDA_SAFE_CALL( hipMemcpy((void *) dmin_d , (void *) &dmin    ,       sizeof(float4), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy((void *) dmax_d , (void *) &dmax    ,       sizeof(float4), hipMemcpyHostToDevice) );
	for(unsigned int idim=0; idim<3; idim++){
		cont='n';
		do{
			if(cont!='n') cout << "Wrong input. Answer with y or n." << endl;
			if(idim==0){
				cout << "X-periodicity (y/n): "; }
			else if(idim==1){
				cout << "Y-periodicity (y/n): ";
			}
			else if(idim==2){
				cout << "Z-periodicity (y/n): ";
			}
			cin >> cont;
		}while(cont!='y' && cont!='n');
		if(cont=='y'){
			per[idim] = true;
			cout << "Updating links ...";
      for(unsigned int i=0; i<nvert; i++)
        newlink_h[i] = -1;
      CUDA_SAFE_CALL( hipMemcpy((void *) newlink, (void *) newlink_h, nvert*sizeof(int)   , hipMemcpyHostToDevice) );
			numBlocks = (int) ceil((float)max(nvert,nbe)/(float)numThreads);
			numBlocks = min(numBlocks,maxblock);

			find_links <<<numBlocks, numThreads>>> (pos_d, nvert, dmax_d, dmin_d, dr, newlink, idim);
			periodicity_links<<<numBlocks,numThreads>>>(pos_d, ep_d, nvert, nbe, dmax_d, dmin_d, dr, newlink, idim);

			CUDA_SAFE_CALL( hipMemcpy((void *) posa,(void *) pos_d, (nvert+nbe)*sizeof(uf4), hipMemcpyDeviceToHost) );
			CUDA_SAFE_CALL( hipMemcpy((void *) ep  ,(void *) ep_d ,         nbe*sizeof(ui4), hipMemcpyDeviceToHost) );
			//if(err!=0) return err;
			//host
			cout << " [OK]" << endl;
		}
	}
	CUDA_SAFE_CALL( hipFree(newlink) );
  delete [] newlink_h;

	//calculate volume of vertex particles
	cout << "\nCalculating volume of vertex particles ...";
	float eps=dr/(float)gres*1e-4;
	int *trisize, *trisize_h;
	float *vol_d;
  bool *per_d;
	trisize_h = new int[nvert];
	for(unsigned int i=0; i<nvert; i++)
		trisize_h[i] = 0;
	CUDA_SAFE_CALL( hipMalloc((void **) &trisize, nvert*sizeof(int  )) );
	CUDA_SAFE_CALL( hipMalloc((void **) &vol_d  , nvert*sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc((void **) &per_d  ,     3*sizeof(bool )) );
	CUDA_SAFE_CALL( hipMemcpy((void *) per_d  , (void *) per      ,     3*sizeof(bool), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy((void *) trisize, (void *) trisize_h, nvert*sizeof(int) , hipMemcpyHostToDevice) );
	numBlocks = (int) ceil((float)nvert/(float)numThreads);
	numBlocks = min(numBlocks,maxblock);
  delete [] trisize_h;

	calc_trisize <<<numBlocks, numThreads>>> (ep_d, trisize, nbe);
#ifndef bdebug
	calc_vert_volume <<<numBlocks, numThreads>>> (pos_d, norm_d, ep_d, vol_d, trisize, dmin_d, dmax_d, nvert, nbe, dr, eps, per_d);
#else
	uf4 *debug, *debug_d;
	int debugs = pow((gres*2+1),3);
	float *debugp, *debugp_d;
	debugp = new float [100];
	debug = new uf4[debugs];
	CUDA_SAFE_CALL( hipMalloc((void **) &debug_d, debugs*sizeof(uf4)) );
	CUDA_SAFE_CALL( hipMalloc((void **) &debugp_d, 100*sizeof(float)) );

	calc_vert_volume <<<numBlocks, numThreads>>> (pos_d, norm_d, ep_d, vol_d, trisize, dmin_d, dmax_d, nvert, nbe, dr, eps, per_d, debug_d, debugp_d);

	CUDA_SAFE_CALL( hipMemcpy((void*) debug, (void*) debug_d, debugs*sizeof(uf4), hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy((void*) debugp, (void*) debugp_d, 100*sizeof(float), hipMemcpyDeviceToHost) );
	for(int i=0; i<30; i++){
		cout << i << " " << debugp[i] << endl;
	}
#endif

	CUDA_SAFE_CALL( hipMemcpy((void *) vola,(void *) vol_d, nvert*sizeof(float), hipMemcpyDeviceToHost) );
	//cout << endl;
	//for(int i=0; i<30; i++) cout << i << " " << vola[i] << endl;
	hipFree( trisize );
	hipFree( vol_d   );

	cout << " [OK]" << endl;

	hipFree(vol_d   );
	hipFree(surf_d  );

	// searching for in/outflow areas
	cout << "\nChecking whether outflow grid is available ...";
	bool boutflow = false;
	int flen = strlen(argv[1]);
	char *cfname;
  cfname = new char[flen+9];
	strncpy(cfname, argv[1], flen-4);
	cfname[flen-4] = '_';
	cfname[flen-3] = 'o';
	cfname[flen-2] = 'u';
	cfname[flen-1] = 't';
	cfname[flen-0] = 'g';
	cfname[flen+1] = 'r';
	cfname[flen+2] = 'i';
	cfname[flen+3] = 'd';
	cfname[flen+8] = '\0';
	strncpy(cfname+flen+4, argv[1]+flen-4, 4);
	stl_file.open(cfname, ios::in);
	if(!stl_file.is_open()){
		boutflow = false;
		cout << " [NO]" << endl;
	}
	else{
		boutflow = true;
		cout << " [YES]" << endl;
    cout << "Checking whether outflow stl file is binary ...";
    bool issolid = true;
    char header[6] = "solid";
    for (int i=0; i<5; i++){
      char dum;
      stl_file.read((char *)&dum, sizeof(char));
      if(dum!=header[i]){
        issolid = false;
        break;
      }
    }
    stl_file.close();
    if(issolid){
      cout << " [NO]" << endl;
      boutflow = false;
    }
    else{
      cout << " [YES]" << endl;
      // reopen file in binary mode
      stl_file.open(cfname, ios::in | ios::binary);
    }
	}
	int outnvert, outnbe;
	uf4 *outposa;
	ui4 *outep;
	if(boutflow){
		// read header
		for (int i=0; i<20; i++){
			float dum;
			stl_file.read((char *)&dum, sizeof(float));
		}
		// get number of facets
		stl_file.read((char *)&num_of_facets, sizeof(int));
		cout << "Reading " << num_of_facets << " facets of outflow geometry ...";

		// define variables
		pos.clear();
		epv.clear();
		for(int i=0;i<3;i++){
      ddum[i] = 0.;
      idum[i] = 0;
		}

		// read data
		through = 0;
		while ((through < num_of_facets) & (!stl_file.eof()))
		{
			for (int i=0; i<12; i++)
			{
				stl_file.read((char *)&m_v_floats[i], sizeof(float));
			}
			for(int j=0;j<3;j++){
				for(int i=0;i<3;i++) ddum[i] = (float)m_v_floats[i+3*(j+1)];
				int k = 0;
				bool found = false;
				for(it = pos.begin(); it < pos.end(); it++){
					float diff = 0;
					for(int i=0;i<3;i++) diff += pow((*it)[i]-ddum[i],2);
					diff = sqrt(diff);
					if(diff < 1e-5*dr){
						idum[j] = k;
						found = true;
						break;
					}
					k++;
				}
				if(!found){
					pos.push_back(ddum);
					idum[j] = k;
				}
			}
			epv.push_back(idum);
			stl_file.read((char *)&attribute, sizeof(short));
			through++;
		}
		stl_file.close();
		if(num_of_facets != epv.size()){
			cout << " [FAILED]" << endl;
			return READ_ERROR;
		}
		outnvert = pos.size();
		outnbe   = epv.size();
		//create and copy vectors to arrays
		outposa  = new uf4   [outnvert];
		outep    = new ui4   [outnbe];
		for(unsigned int i=0; i<max(outnvert,outnbe); i++){
			if(i<outnbe){
				for(int j=0; j<3; j++){
					outep[i].a[j] = epv[i][j];
				}
			}
			if(i<outnvert){
				for(unsigned int j=0; j<3; j++)
					outposa[i].a[j] = pos[i][j];
			}
		}
		pos.clear();
    epv.clear();
    cout << " [OK]" << endl;
	}

	cout << "\nChecking whether inflow grid is available ...";
	bool binflow = false;
	delete [] cfname;
  cfname = new char[flen+8];
	strncpy(cfname, argv[1], flen-4);
	ifstream stl_in_file;
	cfname[flen-4] = '_';
	cfname[flen-3] = 'i';
	cfname[flen-2] = 'n';
	cfname[flen-1] = 'g';
	cfname[flen-0] = 'r';
	cfname[flen+1] = 'i';
	cfname[flen+2] = 'd';
	cfname[flen+7] = '\0';
	strncpy(cfname+flen+3, argv[1]+flen-4, 4);
	stl_in_file.open(cfname, ios::in);
	if(!stl_in_file.is_open()){
		binflow = false;
		cout << " [NO]" << endl;
	}
	else{
		binflow = true;
		cout << " [YES]" << endl;
    cout << "Checking whether inflow stl file is binary ...";
    bool issolid = true;
    char header[6] = "solid";
    for (int i=0; i<5; i++){
      char dum;
      stl_in_file.read((char *)&dum, sizeof(char));
      if(dum!=header[i]){
        issolid = false;
        break;
      }
    }
    stl_in_file.close();
    if(issolid){
      cout << " [NO]" << endl;
      binflow = false;
    }
    else{
      cout << " [YES]" << endl;
      // reopen file in binary mode
      stl_in_file.open(cfname, ios::in | ios::binary);
    }
	}
	int innvert, innbe;
	uf4 *inposa;
	ui4 *inep;
	if(binflow){
		// read header
		for (int i=0; i<20; i++){
			float dum;
			stl_in_file.read((char *)&dum, sizeof(float));
		}
		// get number of facets
		stl_in_file.read((char *)&num_of_facets, sizeof(int));
		cout << "Reading " << num_of_facets << " facets of inflow geometry ...";

		// define variables
		pos.clear();
		epv.clear();
		for(int i=0;i<3;i++){
      ddum[i] = 0.;
      idum[i] = 0;
		}

		// read data
		through = 0;
		while ((through < num_of_facets) & (!stl_in_file.eof()))
		{
			for (int i=0; i<12; i++)
			{
				stl_in_file.read((char *)&m_v_floats[i], sizeof(float));
			}
			for(int j=0;j<3;j++){
				for(int i=0;i<3;i++) ddum[i] = (float)m_v_floats[i+3*(j+1)];
				int k = 0;
				bool found = false;
				for(it = pos.begin(); it < pos.end(); it++){
					float diff = 0;
					for(int i=0;i<3;i++) diff += pow((*it)[i]-ddum[i],2);
					diff = sqrt(diff);
					if(diff < 1e-5*dr){
						idum[j] = k;
						found = true;
						break;
					}
					k++;
				}
				if(!found){
					pos.push_back(ddum);
					idum[j] = k;
				}
			}
			epv.push_back(idum);
			stl_in_file.read((char *)&attribute, sizeof(short));
			through++;
		}
		stl_in_file.close();
		if(num_of_facets != epv.size()){
			cout << " [FAILED]" << endl;
			return READ_ERROR;
		}
		innvert = pos.size();
		innbe   = epv.size();
		//create and copy vectors to arrays
		inposa  = new uf4   [innvert];
		inep    = new ui4   [innbe];
		for(unsigned int i=0; i<max(innvert,innbe); i++){
			if(i<innbe){
				for(int j=0; j<3; j++){
					inep[i].a[j] = epv[i][j];
				}
			}
			if(i<innvert){
				for(unsigned int j=0; j<3; j++)
					inposa[i].a[j] = pos[i][j];
			}
		}
		pos.clear();
    epv.clear();
    cout << " [OK]" << endl;
	}

  /* in/outflow is for version 0.6
	// after reading in data for in/outflow copy data to gpu and identify interior boundary segments
	short *inout;
	if(binflow || boutflow){
		short *inout_d;
    uf4 *outpos_d, *inpos_d;
    ui4 *outep_d , *inep_d;
		inout = new short[nbe];
		CUDA_SAFE_CALL( hipMalloc((void **) &inout_d  ,      nbe*sizeof(short)) );
		CUDA_SAFE_CALL( hipMalloc((void **) &inpos_d  ,  innvert*sizeof(uf4  )) );
		CUDA_SAFE_CALL( hipMalloc((void **) &outpos_d , outnvert*sizeof(uf4  )) );
		CUDA_SAFE_CALL( hipMalloc((void **) &inep_d   ,    innbe*sizeof(uf4  )) );
		CUDA_SAFE_CALL( hipMalloc((void **) &outep_d  ,   outnbe*sizeof(ui4  )) );
		CUDA_SAFE_CALL( hipMemcpy((void *) outposa, (void *) outpos_d, outnvert*sizeof(uf4) , hipMemcpyHostToDevice) );
		CUDA_SAFE_CALL( hipMemcpy((void *) outep  , (void *) outep_d ,   outnbe*sizeof(ui4) , hipMemcpyHostToDevice) );
		CUDA_SAFE_CALL( hipMemcpy((void *) inposa , (void *) inpos_d ,  innvert*sizeof(uf4) , hipMemcpyHostToDevice) );
		CUDA_SAFE_CALL( hipMemcpy((void *) inep   , (void *) inep_d  ,    innbe*sizeof(ui4) , hipMemcpyHostToDevice) );
		numBlocks = (int) ceil((float)nbe/(float)numThreads);
		numBlocks = min(numBlocks,maxblock);

		identifyInOutFlowSegments<<<numBlocks, numThreads>>> (pos_d, nvert, nbe, outpos_d, outep_d, outnbe, inpos_d, inep_d, innbe, eps, inout_d);
	
		CUDA_SAFE_CALL( hipMemcpy((void *) inout_d, (void *) inout, nbe*sizeof(short) , hipMemcpyDeviceToHost) );
		hipFree( inout     );
		hipFree( outpos_d  );
		hipFree( inpos_d );
		hipFree( outep_d   );
		hipFree( inep_d  );
	}
  */

	//setting up fluid particles
	cout << "\nDefining fluid particles ..." << endl;

	cout << "Checking wether coarse grid is available ...";
	bool bcoarse = false;
	strncpy(cfname, argv[1], flen-4);
	cfname[flen-4] = '_';
	cfname[flen-3] = 'c';
	cfname[flen-2] = 'o';
	cfname[flen-1] = 'a';
	cfname[flen-0] = 'r';
	cfname[flen+1] = 's';
	cfname[flen+2] = 'e';
	cfname[flen+7] = '\0';
	strncpy(cfname+flen+3, argv[1]+flen-4, 4);
	stl_file.open(cfname, ios::in);
	if(!stl_file.is_open()){
		bcoarse = false;
		cout << " [NO]" << endl;
	}
	else{
		bcoarse = true;
		cout << " [YES]" << endl;
    cout << "Checking whether coarse geometry stl file is binary ...";
    bool issolid = true;
    char header[6] = "solid";
    for (int i=0; i<5; i++){
      char dum;
      stl_file.read((char *)&dum, sizeof(char));
      if(dum!=header[i]){
        issolid = false;
        break;
      }
    }
    stl_file.close();
    if(issolid){
      cout << " [NO]" << endl;
      bcoarse = false;
    }
    else{
      cout << " [YES]" << endl;
      // reopen file in binary mode
      stl_file.open(cfname, ios::in | ios::binary);
    }
	}

	cout << "Checking wether fluid geometry is available ...";
	bool bfgeom = false;
	strncpy(cfname, argv[1], flen-4);
	cfname[flen-4] = '_';
	cfname[flen-3] = 'f';
	cfname[flen-2] = 's';
	cfname[flen-1] = 'h';
	cfname[flen-0] = 'a';
	cfname[flen+1] = 'p';
	cfname[flen+2] = 'e';
	strncpy(cfname+flen+3, argv[1]+flen-4, 4);

	ifstream fstl_file (cfname, ios::in);
	if(!fstl_file.is_open()){
		bfgeom = false;
		cout << " [NO]" << endl;
	}
	else{
		bfgeom = true;
		cout << " [YES]" << endl;
    cout << "Checking whether fluid geometry stl file is binary ...";
    bool issolid = true;
    char header[6] = "solid";
    for (int i=0; i<5; i++){
      char dum;
      fstl_file.read((char *)&dum, sizeof(char));
      if(dum!=header[i]){
        issolid = false;
        break;
      }
    }
    fstl_file.close();
    if(issolid){
      cout << " [NO]" << endl;
      bfgeom = false;
    }
    else{
      cout << " [YES]" << endl;
      // reopen file in binary mode
      fstl_file.open(cfname, ios::in | ios::binary);
      if(!fstl_file.is_open()){
        cout << "Error: could not reopen fluid geometry file in binary mode" << endl;
        return -1;
      }
    }
	}
  delete [] cfname;

	bool set = true;
	bool firstfgeom = true;
	unsigned int cnvert, cnbe;
	uf4 *cnorma, *cposa;
	ui4 *cep;
	unsigned int nfluid = 0;
	unsigned int nfbox = 0;
	unsigned int maxf = 0, maxfn;
	int opt;
	unsigned int *fpos, *fpos_d;
  unsigned int *nfi_d;

	eps = 1e-10;
	for(unsigned int i=0; i<3; i++)
		eps = max((dmax.a[i]-dmin.a[i])*1e-6,eps);
	cont = 'n';
	do{
		if(cont!='n') cout << "Wrong input. Answer with y or n." << endl;
		cout << "Specify fluid container (y/n): ";
		cin >> cont;
		if(cont=='n') set = false;
	}while(cont!='y' && cont!='n');

  if(set){
    cout << "Specify fluid container:" << endl;
    cout << "Min coordinates (x,y,z): ";
    // From here on dmin, dmax represent the fluid container and no longer the domain container.
    cin >> dmin.a[0] >> dmin.a[1] >> dmin.a[2];
    cout << "Max coordinates (x,y,z): ";
    cin >> dmax.a[0] >> dmax.a[1] >> dmax.a[2];
    CUDA_SAFE_CALL( hipMemcpy((void *) dmin_d , (void *) &dmin    ,       sizeof(float4), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy((void *) dmax_d , (void *) &dmax    ,       sizeof(float4), hipMemcpyHostToDevice) );
  }
  else{
    cout << "Using whole geometry as fluid container." << endl;
  }

	maxfn = (int)floor((dmax.a[0]-dmin.a[0]+eps)/dr+1)*floor((dmax.a[1]-dmin.a[1]+eps)/dr+1)*floor((dmax.a[2]-dmin.a[2]+eps)/dr+1);
	maxf = (int)ceil(float(maxfn)/8./((float)sizeof(unsigned int)));
	fpos = new unsigned int [maxf];
	CUDA_SAFE_CALL( hipMalloc((void **) &fpos_d, maxf*sizeof(unsigned int)) );
  CUDA_SAFE_CALL( hipMalloc((void **) &nfi_d, sizeof(unsigned int)) );
  for(unsigned int i=0; i<maxf; i++) fpos[i] = 0;
  CUDA_SAFE_CALL( hipMemcpy((void *) fpos_d, (void *) fpos, maxf*sizeof(unsigned int), hipMemcpyHostToDevice) );

  set = true;
	while(set){
		xmin = xmax = ymin = ymax = zmin = zmax = 0.;
		if(bfgeom){
			cout << "Choose option:" << endl;
			cout << " 1 ... Fluid in a box" << endl;
			cout << " 2 ... Fluid based on geometry" << endl;
			cout << "Input: ";
			opt = 0;
			cin >> opt;
			while(opt<1 || opt>2){
				cout << "Wrong input try again: ";
				cin >> opt;
			}
		}
		else{
			opt = 1;
		}

    // data for geometry bounding grid and fluid bounding grid
    unsigned int fnvert=0, fnbe=0;
    uf4 *fposa=NULL, *fnorma=NULL;
    ui4 *fep=NULL;

		if(opt==1){ // fluid based on rectangular box
			cout << "Enter dimensions of fluid box:" << endl;
			cout << "xmin xmax: ";
			cin >> xmin >> xmax;
			cout << "ymin ymax: ";
			cin >> ymin >> ymax;
			cout << "zmin zmax: ";
			cin >> zmin >> zmax;
			if(fabs(xmin-xmax)<1e-5*dr || fabs(ymin-ymax)<1e-5*dr || fabs(zmin-zmax)<1e-5*dr){
				cout << "\nMistake in input for fluid box dimensions" << endl;
				cout << "Fluid particle definition ... [FAILED]" << endl;
				return FLUID_NDEF;
			}
			numBlocks = (int) ceil((float)maxf/(float)numThreads);
			numBlocks = min(numBlocks,maxblock);

			Lock lock_f;
      unsigned int nfi=0;
      CUDA_SAFE_CALL( hipMemcpy((void *) nfi_d, (void *) &nfi, sizeof(unsigned int), hipMemcpyHostToDevice) );

			fill_fluid<<<numBlocks, numThreads>>> (fpos_d, nfi_d, xmin, xmax, ymin, ymax, zmin, zmax, dmin_d, dmax_d, eps, dr, lock_f);

			CUDA_SAFE_CALL( hipMemcpy((void *) &nfi, (void *) nfi_d, sizeof(unsigned int), hipMemcpyDeviceToHost) );
			nfluid += nfi;
				
		}

		else if(opt==2){ // fluid based on geometry
			// get seed point
			float spos[3], dr_wall;
			cout << "Please specify a seed point." << endl;
			cout << "x, y, z = ";
			cin >> spos[0] >> spos[1] >> spos[2];
      cout << "Specify distance from fluid particles to vertex particles and segments: ";
      cin >> dr_wall;
      // initialize placing of seed point
      int ispos = (int)round((spos[0]-dmin.a[0]+eps)/dr);
      int jspos = (int)round((spos[1]-dmin.a[1]+eps)/dr);
      int kspos = (int)round((spos[2]-dmin.a[2]+eps)/dr);
      int idimg = (int)floor((dmax.a[0]-dmin.a[0]+eps)/dr+1);
      int jdimg = (int)floor((dmax.a[1]-dmin.a[1]+eps)/dr+1);
      int sInd = ispos + jspos*idimg + kspos*idimg*jdimg;
      int sIndex = sInd/bitPerUint;
      unsigned int sBit = 1<<(sInd%bitPerUint);

			// initialize geometry if first run
			if(firstfgeom){
				firstfgeom = false;

				hipFree(norm_d  );
				hipFree(pos_d   );
				hipFree(ep_d    );

				// if coarse grid for geometry is available read it
				if(bcoarse){
					// read header
					for (int i=0; i<20; i++){
						float dum;
						stl_file.read((char *)&dum, sizeof(float));
					}
					// get number of facets
					stl_file.read((char *)&num_of_facets, sizeof(int));
					cout << "Reading " << num_of_facets << " facets of coarse geometry ...";

					// define variables
					pos.clear();
					norm.clear();
					epv.clear();
					for(int i=0;i<3;i++){
            ddum[i] = 0.;
            idum[i] = 0;
					}

					// read data
					through = 0;
					while ((through < num_of_facets) & (!stl_file.eof()))
					{
						for (int i=0; i<12; i++)
						{
							stl_file.read((char *)&m_v_floats[i], sizeof(float));
						}
						for(int i=0;i<3;i++) ddum[i] = (float)m_v_floats[i];
						norm.push_back(ddum);
						for(int j=0;j<3;j++){
							for(int i=0;i<3;i++) ddum[i] = (float)m_v_floats[i+3*(j+1)];
							int k = 0;
							bool found = false;
							for(it = pos.begin(); it < pos.end(); it++){
								float diff = 0;
								for(int i=0;i<3;i++) diff += pow((*it)[i]-ddum[i],2);
								diff = sqrt(diff);
								if(diff < 1e-5*dr){
									idum[j] = k;
									found = true;
									break;
								}
								k++;
							}
							if(!found){
								pos.push_back(ddum);
								idum[j] = k;
							}
						}
						epv.push_back(idum);
						stl_file.read((char *)&attribute, sizeof(short));
						through++;
					}
					stl_file.close();
					if(num_of_facets != norm.size()){
						cout << " [FAILED]" << endl;
						return READ_ERROR;
					}
					fnvert = pos.size();
					fnbe   = norm.size();
					//create and copy vectors to arrays
					fnorma = new uf4   [fnbe];
					fposa  = new uf4   [fnvert];
					fep    = new ui4   [fnbe];
					for(unsigned int i=0; i<max(fnvert,fnbe); i++){
						if(i<fnbe){
							for(int j=0; j<3; j++){
								fnorma[i].a[j] = norm[i][j];
								fep[i].a[j] = epv[i][j];
							}
						}
						if(i<fnvert){
							for(unsigned int j=0; j<3; j++)
								fposa[i].a[j] = pos[i][j];
						}
					}
					pos.clear();
          epv.clear();
					norm.clear();
          cout << " [OK]" << endl;
				}
        else{
          // no coarse geometry available, copy fine one to f* arrays
          fnvert = nvert;
          fnbe = nbe;
          fep = new ui4 [fnbe];
          fnorma = new uf4 [fnbe];
          fposa = new uf4 [fnvert];
          for(unsigned int i=0; i<max(fnvert,fnbe); i++){
            if(i<fnbe){
              fep[i] = ep[i];
              fnorma[i] = norma[i];
            }
            if(i<fnvert)
              fposa[i] = posa[i];
          }
        }

				// read fluid geometry
				// read header
				for (int i=0; i<20; i++){
				float dum;
				fstl_file.read((char *)&dum, sizeof(float));
				}
				// get number of facets
				fstl_file.read((char *)&num_of_facets, sizeof(int));
				cout << "Reading " << num_of_facets << " facets of fluid geometry ...";

				// define variables
				pos.clear();
				norm.clear();
				epv.clear();
				for(int i=0;i<3;i++){
					ddum[i] = 0.;
					idum[i] = 0;
				}

				// read data
				through = 0;
				while ((through < num_of_facets) & (!fstl_file.eof()))
				{
					for (int i=0; i<12; i++){
						fstl_file.read((char *)&m_v_floats[i], sizeof(float));
					}
					for(int i=0;i<3;i++) ddum[i] = (float)m_v_floats[i];
					norm.push_back(ddum);
					for(int j=0;j<3;j++){
						for(int i=0;i<3;i++) ddum[i] = (float)m_v_floats[i+3*(j+1)];
						int k = 0;
						bool found = false;
						for(it = pos.begin(); it < pos.end(); it++){
							float diff = 0;
							for(int i=0;i<3;i++) diff += pow((*it)[i]-ddum[i],2);
							diff = sqrt(diff);
							if(diff < 1e-5*dr){
								idum[j] = k+fnvert;
								found = true;
								break;
							}
							k++;
						}
						if(!found){
							pos.push_back(ddum);
							idum[j] = k+fnvert;
						}
					}
					epv.push_back(idum);
					fstl_file.read((char *)&attribute, sizeof(short));
					through++;
				}
				fstl_file.close();
				if(num_of_facets != norm.size()){
					cout << " [FAILED]" << endl;
					return READ_ERROR;
				}
				cnvert = pos.size();
				cnbe   = norm.size();
        cout << " [OK]" << endl;
        cout << "Merging arrays and preparing device for filling ...";
				//create and copy vectors to arrays
				cnorma = new uf4   [fnbe];
				cposa  = new uf4   [fnvert];
				cep    = new ui4   [fnbe];
				for(unsigned int i=0; i<max(fnbe,fnvert); i++){
					if(i<fnbe){
						cnorma[i] = fnorma[i];
						cep   [i] = fep   [i];
					}
					if(i<fnvert){
						cposa [i] = fposa [i];
					}
				}
				delete [] fnorma;
				delete [] fposa;
				delete [] fep;
				fnorma = new uf4   [fnbe+cnbe];
				fposa  = new uf4   [fnvert+cnvert];
				fep    = new ui4   [fnbe+cnbe];
				for(unsigned int i=0; i<max(fnbe,fnvert); i++){
					if(i<fnbe){
						fnorma[i] = cnorma[i];
						fep   [i] = cep   [i];
					}
					if(i<fnvert){
						fposa [i] = cposa [i];
					}
				}
				delete [] cnorma;
				delete [] cposa;
				delete [] cep;
				for(unsigned int i=0; i<max(cnvert,cnbe); i++){
					if(i<cnbe){
						for(int j=0; j<3; j++){
							fnorma[i+fnbe].a[j] = norm[i][j];
							fep[i+fnbe].a[j] = epv[i][j];
						}
					}
					if(i<cnvert){
						for(int j=0; j<3; j++)
							fposa[i+fnvert].a[j] = pos[i][j];
					}
				}
				fnvert += cnvert;
				fnbe += cnbe;
				pos.clear();
				epv.clear();
				norm.clear();
				CUDA_SAFE_CALL( hipMalloc((void **) &norm_d,   fnbe*sizeof(uf4  )) );
				CUDA_SAFE_CALL( hipMalloc((void **) &pos_d , fnvert*sizeof(uf4  )) );
				CUDA_SAFE_CALL( hipMalloc((void **) &ep_d  ,   fnbe*sizeof(ui4  )) );
				CUDA_SAFE_CALL( hipMemcpy((void *) norm_d, (void *) fnorma,   fnbe*sizeof(uf4), hipMemcpyHostToDevice) );
				CUDA_SAFE_CALL( hipMemcpy((void *) pos_d , (void *) fposa , fnvert*sizeof(uf4), hipMemcpyHostToDevice) );
				CUDA_SAFE_CALL( hipMemcpy((void *) ep_d  , (void *) fep   ,   fnbe*sizeof(ui4), hipMemcpyHostToDevice) );
        
        numBlocks = (int) ceil((float)maxf/(float)numThreads);
        numBlocks = min(numBlocks,maxblock);
        cout << " [OK]" << endl;
			} // end firstfgeom

      unsigned int nfi;
      unsigned int iteration = 0;
      do{
        Lock lock_f;
        iteration++;
        nfi = 0;
        CUDA_SAFE_CALL( hipMemcpy((void *) nfi_d, (void *) &nfi, sizeof(unsigned int), hipMemcpyHostToDevice) );

        fill_fluid_complex<<<numBlocks, numThreads>>> (fpos_d, nfi_d, norm_d, ep_d, pos_d, fnbe, dmin_d, dmax_d, eps, dr, sIndex, sBit, lock_f, bcoarse, cnbe, dr_wall);

        CUDA_SAFE_CALL( hipMemcpy((void *) &nfi, (void *) nfi_d, sizeof(unsigned int), hipMemcpyDeviceToHost) );
        nfluid += nfi;
      } while(nfi > 0 && iteration < max_iterations);
		}

		cont = 'n';
		do{
			if(cont!='n') cout << "Wrong input. Answer with y or n." << endl;
			cout << "Another fluid container (y/n): ";
			cin >> cont;
      if(nfbox==maxfbox){
        cont = 'n';
        cout << "Maximum number of fluid boxes reached, no more fluid can be added." << endl;
      }
			if(cont=='n') set = false;
		}while(cont!='y' && cont!='n');

    if(!firstfgeom && cont == 'n'){
      delete [] fposa;
      delete [] fnorma;
      delete [] fep;
    }
	}
	CUDA_SAFE_CALL( hipMemcpy((void *) fpos, (void *) fpos_d, maxf*sizeof(unsigned int), hipMemcpyDeviceToHost) );
	cout << "\nCreation of " << nfluid << " fluid particles completed. [OK]" << endl;
	hipFree( fpos_d );
	hipFree( nfi_d  );
	hipFree( norm_d );
	hipFree( pos_d  );
	hipFree( ep_d   );

	//prepare output structure for particles
	cout << "Creating and initializing of output buffer of particles ...";
	OutBuf *buf;
#ifndef bdebug
	unsigned int nelem = nvert+nbe+nfluid;
#else
	unsigned int nelem = nvert+nbe+nfluid+debugs;
#endif
	buf = new OutBuf[nelem];
	int k=0;
	unsigned int m,n,imin[3];
  float fluid_vol = pow(dr,3);
	imin[0] = int(floor((dmax.a[0]-dmin.a[0]+eps)/dr))+1;
	imin[1] = int(floor((dmax.a[1]-dmin.a[1]+eps)/dr))+1;
	imin[2] = int(floor((dmax.a[2]-dmin.a[2]+eps)/dr))+1;
	//free particles
	for(unsigned int j=0; j<maxfn; j++){
		int i = j/bitPerUint;
		int l = j%bitPerUint;
		m = 1 << l;
		if(fpos[i] & m){
			m = j/(imin[1]*imin[0]);
			buf[k].z = dmin.a[2]+dr*(float)m;
			n = j%(imin[1]*imin[0]);
			m = n/imin[0];
			buf[k].y = dmin.a[1]+dr*(float)m;
			m = n%imin[0];
			buf[k].x = dmin.a[0]+dr*(float)m;
			buf[k].nx = 0.;
			buf[k].ny = 0.;
			buf[k].nz = 0.;
			buf[k].vol = fluid_vol;
			buf[k].surf = 0.;
			buf[k].kpar = 1;
			buf[k].kfluid = 1;
			buf[k].kent = 1;
			buf[k].kparmob = 0;
			buf[k].iref = k;
			buf[k].ep1 = 0;
			buf[k].ep2 = 0;
			buf[k].ep3 = 0;
			k++;
		}
	}
	//vertex particles
  int *nvshift;
  nvshift = new int[nvert];
  for(unsigned int i=0; i<nvert; i++)
    nvshift[i] = 0;
  int ishift = 0;
	for(unsigned int i=0; i<nvert; i++){
		if(posa[i].a[0] < -1e9){
			nelem--;
      ishift++;
			continue;
		}
    nvshift[i] = ishift;
		buf[k].x = posa[i].a[0];
		buf[k].y = posa[i].a[1];
		buf[k].z = posa[i].a[2];
		buf[k].nx = 0.;
		buf[k].ny = 0.;
		buf[k].nz = 0.;
		buf[k].vol = vola[i];
		buf[k].surf = 0.;
		buf[k].kpar = 2;
		buf[k].kfluid = 1;
		buf[k].kent = 1;
		buf[k].kparmob = 0;
		buf[k].iref = k;
		buf[k].ep1 = 0;
		buf[k].ep2 = 0;
		buf[k].ep3 = 0;
		k++;
	}
	//boundary segments
	for(unsigned int i=nvert; i<nvert+nbe; i++){
		buf[k].x = posa[i].a[0];
		buf[k].y = posa[i].a[1];
		buf[k].z = posa[i].a[2];
		buf[k].nx = norma[i-nvert].a[0];
		buf[k].ny = norma[i-nvert].a[1];
		buf[k].nz = norma[i-nvert].a[2];
		buf[k].vol = 0.;
		buf[k].surf = surf[i-nvert];
		buf[k].kpar = 3;
    /* in/outflow is for version 0.6
		if(binflow || boutflow)
			buf[k].kpar += inout[i];
    */
		buf[k].kfluid = 1;
		buf[k].kent = 1;
		buf[k].kparmob = 0;
		buf[k].iref = k;
		buf[k].ep1 = nfluid+ep[i-nvert].a[0] - nvshift[ep[i-nvert].a[0]];
		buf[k].ep2 = nfluid+ep[i-nvert].a[1] - nvshift[ep[i-nvert].a[1]];
		buf[k].ep3 = nfluid+ep[i-nvert].a[2] - nvshift[ep[i-nvert].a[2]];
		k++;
	}
  delete [] nvshift;
#ifdef bdebug
	//debug
	for(unsigned int i=0; i<debugs; i++){
		buf[k].x = debug[i].a[0];
		buf[k].y = debug[i].a[1];
		buf[k].z = debug[i].a[2];
		buf[k].nx = 0;
		buf[k].ny = 0;
		buf[k].nz = 0;
		buf[k].vol = debug[i].a[3];
		buf[k].surf = 0.;
		buf[k].kpar = 4;
		buf[k].kfluid = 1;
		buf[k].kent = 1;
		buf[k].kparmob = 0;
		buf[k].iref = k;
		buf[k].ep1 = 0;
		buf[k].ep2 = 0;
		buf[k].ep3 = 0;
		k++;
	}
#endif
	cout << " [OK]" << endl;

	//Output of particles
	char *fname = new char[flen+5];
	const char *fend = "h5sph";
	float time = 0.;
	fname[0] = '0';
	fname[1] = '.';
	strncpy(fname+2, argv[1], flen-3);
	strncpy(fname+flen-1, fend, strlen(fend));
	fname[flen+4] = '\0';
	cout << "Writing output to file " << fname << " ...";
	int err = hdf5_output( buf, nelem, fname, &time);
	if(err==0){ cout << " [OK]" << endl; }
	else {
		cout << " [FAILED]" << endl;
		return WRITE_FAIL;
	}

	//Free memory
	//Arrays
	delete [] norma;
	delete [] posa;
	delete [] vola;
	delete [] surf;
	delete [] ep;
	delete [] buf;
	delete [] fname;
	delete [] fpos;
	//Cuda
	hipFree( per_d   );
	hipFree( dmin_d  );
	hipFree( dmax_d  );

	//End
	return 0;
}
#endif
